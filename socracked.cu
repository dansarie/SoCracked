#include "hip/hip_runtime.h"
/* socracked.cu

   CUDA bitslice implementation of attacks on SoCracked.

   Copyright (C) 2017-2018 Marcus Dansarie <marcus@dansarie.se>

   This program is free software: you can redistribute it and/or modify
   it under the terms of the GNU General Public License as published by
   the Free Software Foundation, either version 3 of the License, or
   (at your option) any later version.

   This program is distributed in the hope that it will be useful,
   but WITHOUT ANY WARRANTY; without even the implied warranty of
   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
   GNU General Public License for more details.

   You should have received a copy of the GNU General Public License
   along with this program. If not, see <http://www.gnu.org/licenses/>. */

#include <assert.h>
#include <stdio.h>
#include <stdint.h>
#include <unistd.h>

#include <hip/hip_runtime_api.h>

#include "sodark.h"
#include "socracked.h"
#include "socracked_cuda.h"

typedef struct {
  int b0;
  int b1;
  int b2;
  int b3;
  int b4;
  int b5;
  int b6;
  int b7;
} eightbits;

typedef struct {
  eightbits a;
  eightbits b;
  eightbits c;
} twentyfourbits;

/* Device constant memory. */

__constant__ int pt1_c[24];
__constant__ int pt2_c[24];
__constant__ int ct1_c[24];
__constant__ int tw1_c[64];
__constant__ int tw2_c[64];
__constant__ int key_c[24];
__constant__ int key3_c[256];

/* Device functions */

/* Macro for using the CUDA lop3.b32 instruction.
   a - output variable
   b - input 1
   c - input 2
   d - input 3
   e - lookup table. */
#define LUT(a,b,c,d,e) asm("lop3.b32 %0, %1, %2, %3, "#e";" : "=r"(a): "r"(b), "r"(c), "r"(d));

/* The functions s0 to s7 calculate one output bit each of the SoDark S-box. */

__device__ __forceinline__ int s0(eightbits in) {
  int var8;  LUT(var8, in.b6, in.b3, in.b4, 0x5e);
  int var9 = var8 | in.b5;
  int var10; LUT(var10, in.b5, in.b4, in.b6, 0x8e);
  int var11; LUT(var11, in.b1, var9, var10, 0xac);
  int var12; LUT(var12, in.b3, in.b6, var10, 0x68);
  int var13; LUT(var13, in.b1, in.b4, var11, 0x8f);
  int var14; LUT(var14, var13, var12, in.b5, 0xca);
  int var15; LUT(var15, in.b2, var11, var14, 0xac);
  int var16; LUT(var16, in.b2, in.b4, var13, 0x7c);
  int var17; LUT(var17, in.b1, in.b6, var14, 0x7a);
  int var18; LUT(var18, var17, var16, var9, 0x96);
  int var19; LUT(var19, in.b2, var12, var18, 0x09);
  int var20; LUT(var20, in.b1, in.b4, in.b5, 0x25);
  int var21; LUT(var21, var20, var19, in.b6, 0xed);
  int var22; LUT(var22, in.b3, var18, var21, 0xac);
  int var23; LUT(var23, in.b7, var15, var22, 0xac);
  int var24; LUT(var24, in.b3, var16, var20, 0x63);
  int var25; LUT(var25, var24, var12, var15, 0x09);
  int var26; LUT(var26, in.b4, var25, var21, 0x71);
  int var27; LUT(var27, in.b6, var25, var26, 0xac);
  int var28; LUT(var28, in.b5, in.b6, var25, 0x6b);
  int var29; LUT(var29, var28, var8, var14, 0x58);
  int var30; LUT(var30, in.b1, var27, var29, 0xac);
  int var31; LUT(var31, in.b2, var23, var24, 0x4e);
  int var32; LUT(var32, var31, var17, var21, 0x92);
  int var33; LUT(var33, in.b2, var10, var30, 0x42);
  int var34; LUT(var34, var33, var13, var19, 0x41);
  int var35; LUT(var35, in.b6, var32, var34, 0xac);
  int var36; LUT(var36, in.b7, var30, var35, 0xac);
  int out0;  LUT(out0, in.b0, var23, var36, 0xac);
  return out0;
}

__device__ __forceinline__ int s1(eightbits in) {
  int var8;  LUT(var8, in.b4, in.b5, in.b6, 0x76);
  int var9;  LUT(var9, in.b5, in.b6, in.b7, 0xef);
  int var10; LUT(var10, in.b4, var8, var9, 0xac);
  int var11; LUT(var11, in.b4, in.b5, in.b6, 0x18);
  int var12; LUT(var12, var11, in.b7, var8, 0xa9);
  int var13; LUT(var13, in.b3, var10, var12, 0xac);
  int var14; LUT(var14, in.b7, var9, var12, 0x86);
  int var15; LUT(var15, in.b3, in.b4, var13, 0xe3);
  int var16; LUT(var16, var15, var14, var11, 0x8b);
  int var17; LUT(var17, in.b1, var13, var16, 0xac);
  int var18; LUT(var18, in.b4, in.b6, var17, 0x49);
  int var19; LUT(var19, in.b3, in.b7, var11, 0x36);
  int var20; LUT(var20, var19, var18, var14, 0xbc);
  int var21; LUT(var21, in.b6, var15, var20, 0x96);
  int var22; LUT(var22, in.b3, var11, var13, 0x29);
  int var23; LUT(var23, var22, var21, var14, 0x9e);
  int var24; LUT(var24, in.b1, var20, var23, 0xac);
  int var25; LUT(var25, in.b2, var17, var24, 0xac);
  int var26; LUT(var26, in.b3, in.b6, var24, 0x7a);
  int var27; LUT(var27, in.b1, var9, var21, 0x60);
  int var28; LUT(var28, var27, var26, var16, 0x1c);
  int var29; LUT(var29, in.b1, var8, var28, 0x97);
  int var30; LUT(var30, var29, in.b6, var23, 0xd6);
  int var31; LUT(var31, in.b4, var28, var30, 0xac);
  int var32; LUT(var32, var21, var24, var29, 0xc5);
  int var33; LUT(var33, var12, var15, var19, 0x78);
  int var34; LUT(var34, var33, var32, in.b7, 0xd6);
  int var35; LUT(var35, var10, var14, var18, 0x7a);
  int var36; LUT(var36, var35, in.b3, var29, 0x87);
  int var37; LUT(var37, in.b1, var34, var36, 0xac);
  int var38; LUT(var38, in.b2, var31, var37, 0xac);
  int out1;  LUT(out1, in.b0, var25, var38, 0xac);
  return out1;
}

__device__ __forceinline__ int s2(eightbits in) {
  int var8;  LUT(var8, in.b5, in.b7, in.b6, 0xe5);
  int var9;  LUT(var9, in.b6, in.b7, in.b5, 0xc7);
  int var10; LUT(var10, in.b3, var8, var9, 0xac);
  int var11; LUT(var11, in.b3, in.b6, in.b7, 0x2c);
  int var12; LUT(var12, var11, var8, var9, 0xe3);
  int var13; LUT(var13, in.b0, var10, var12, 0xac);
  int var14; LUT(var14, in.b6, var10, var13, 0xa7);
  int var15; LUT(var15, in.b0, in.b3, var9, 0x86);
  int var16; LUT(var16, var15, var14, in.b7, 0x5c);
  int var17; LUT(var17, in.b4, var13, var16, 0xac);
  int var18; LUT(var18, in.b7, var15, var17, 0xd6);
  int var19; LUT(var19, in.b4, in.b5, var9, 0xd4);
  int var20; LUT(var20, var19, var18, var16, 0xd3);
  int var21; LUT(var21, in.b4, var16, var20, 0x8e);
  int var22; LUT(var22, var21, in.b3, var19, 0x69);
  int var23; LUT(var23, in.b0, var20, var22, 0xac);
  int var24; LUT(var24, in.b1, var17, var23, 0xac);
  int var25; LUT(var25, in.b6, var8, var21, 0x38);
  int var26; LUT(var26, in.b4, var11, var22, 0x97);
  int var27; LUT(var27, var26, var25, var20, 0xc2);
  int var28; LUT(var28, var11, var21, var24, 0xa6);
  int var29; LUT(var29, var10, var22, var25, 0x6f);
  int var30; LUT(var30, var29, var28, in.b7, 0x38);
  int var31; LUT(var31, in.b0, var27, var30, 0xac);
  int var32; LUT(var32, var10, var14, var25, 0x58);
  int var33; LUT(var33, var32, var26, var30, 0xb4);
  int var34; LUT(var34, in.b7, var8, var24, 0x3a);
  int var35; LUT(var35, in.b5, var9, var33, 0x52);
  int var36; LUT(var36, var35, var34, in.b4, 0x79);
  int var37; LUT(var37, in.b0, var33, var36, 0xac);
  int var38; LUT(var38, in.b1, var31, var37, 0xac);
  int out2;  LUT(out2, in.b2, var24, var38, 0xac);
  return out2;
}

__device__ __forceinline__ int s3(eightbits in) {
  int var8;  LUT(var8, in.b3, in.b2, in.b1, 0x85);
  int var9;  LUT(var9, in.b4, var8, in.b5, 0xac);
  int var10; LUT(var10, in.b2, in.b3, in.b4, 0xad);
  int var11; LUT(var11, var10, in.b1, var8, 0x9c);
  int var12; LUT(var12, in.b5, var9, var11, 0xac);
  int var13; LUT(var13, in.b5, var10, var11, 0xd9);
  int var14; LUT(var14, in.b4, var8, var12, 0x14);
  int var15; LUT(var15, var14, var13, in.b2, 0xa6);
  int var16; LUT(var16, in.b7, var12, var15, 0xac);
  int var17; LUT(var17, in.b5, var14, var16, 0x1a);
  int var18; LUT(var18, in.b4, in.b7, var10, 0x75);
  int var19; LUT(var19, var18, var17, var15, 0x25);
  int var20; LUT(var20, var9, var10, var15, 0xe6);
  int var21; LUT(var21, in.b1, in.b2, in.b5, 0x6d);
  int var22; LUT(var22, var21, var20, in.b7, 0xb4);
  int var23; LUT(var23, in.b3, var19, var22, 0xac);
  int var24; LUT(var24, in.b6, var16, var23, 0xac);
  int var25; LUT(var25, in.b6, var18, var21, 0x18);
  int var26; LUT(var26, in.b2, in.b5, var11, 0xe1);
  int var27; LUT(var27, var26, var25, var23, 0xde);
  int var28; LUT(var28, var8, var13, var24, 0xd4);
  int var29; LUT(var29, in.b6, var10, var22, 0xc4);
  int var30; LUT(var30, var29, var28, var14, 0x96);
  int var31; LUT(var31, in.b1, var27, var30, 0xac);
  int var32; LUT(var32, var17, var21, var31, 0x5b);
  int var33; LUT(var33, var32, in.b5, var29, 0x61);
  int var34; LUT(var34, var19, var31, var33, 0x92);
  int var35; LUT(var35, in.b4, in.b5, var25, 0x5c);
  int var36; LUT(var36, var35, var34, var27, 0x68);
  int var37; LUT(var37, in.b3, var33, var36, 0xac);
  int var38; LUT(var38, in.b7, var31, var37, 0xac);
  int out3;  LUT(out3, in.b0, var24, var38, 0xac);
  return out3;
}

__device__ __forceinline__ int s4(eightbits in) {
  int var8;  LUT(var8, in.b2, in.b5, in.b1, 0xe3);
  int var9;  LUT(var9, var8, in.b1, in.b7, 0xbe);
  int var10; LUT(var10, in.b0, var8, var9, 0xac);
  int var11; LUT(var11, var9, in.b1, in.b2, 0x49);
  int var12; LUT(var12, in.b4, var10, var11, 0xac);
  int var13; LUT(var13, var8, var9, var12, 0x6f);
  int var14; LUT(var14, in.b0, in.b4, in.b7, 0x71);
  int var15; LUT(var15, var14, var13, in.b2, 0x68);
  int var16; LUT(var16, in.b6, var12, var15, 0xac);
  int var17; LUT(var17, in.b4, in.b6, var13, 0x68);
  int var18; LUT(var18, in.b0, in.b1, var16, 0x27);
  int var19; LUT(var19, var18, var17, var15, 0x79);
  int var20; LUT(var20, var14, var18, var19, 0xf6);
  int var21; LUT(var21, in.b0, in.b1, var13, 0x92);
  int var22; LUT(var22, var21, var20, in.b6, 0x69);
  int var23; LUT(var23, in.b7, var19, var22, 0xac);
  int var24; LUT(var24, in.b3, var16, var23, 0xac);
  int var25; LUT(var25, in.b7, var19, var22, 0x26);
  int var26; LUT(var26, in.b1, in.b6, var15, 0xd1);
  int var27; LUT(var27, var26, var25, var17, 0x9c);
  int var28; LUT(var28, var12, var18, var20, 0xe0);
  int var29; LUT(var29, var10, var22, var23, 0x29);
  int var30; LUT(var30, var29, var28, in.b4, 0x96);
  int var31; LUT(var31, in.b3, var27, var30, 0xac);
  int var32; LUT(var32, in.b6, var24, var28, 0x56);
  int var33; LUT(var33, var32, in.b1, in.b3, 0x79);
  int var34; LUT(var34, var21, var30, var33, 0x84);
  int var35; LUT(var35, in.b1, in.b7, var32, 0xa9);
  int var36; LUT(var36, var35, var34, in.b6, 0x9a);
  int var37; LUT(var37, in.b0, var33, var36, 0xac);
  int var38; LUT(var38, in.b2, var31, var37, 0xac);
  int out4;  LUT(out4, in.b5, var24, var38, 0xac);
  return out4;
}

__device__ __forceinline__ int s5(eightbits in) {
  int var8;  LUT(var8, in.b7, in.b5, in.b2, 0xa8);
  int var9;  LUT(var9, in.b5, in.b3, in.b2, 0x3d);
  int var10; LUT(var10, in.b0, var8, var9, 0xac);
  int var11; LUT(var11, in.b0, in.b3, in.b5, 0xd3);
  int var12; LUT(var12, var11, var8, var9, 0x92);
  int var13; LUT(var13, in.b1, var10, var12, 0xac);
  int var14; LUT(var14, in.b3, var8, var11, 0x86);
  int var15; LUT(var15, in.b1, var9, var12, 0x14);
  int var16; LUT(var16, var15, var14, var13, 0x79);
  int var17; LUT(var17, in.b7, var13, var16, 0xac);
  int var18; LUT(var18, in.b0, in.b5, var12, 0x5a);
  int var19; LUT(var19, var18, in.b7, var9, 0x94);
  int var20; LUT(var20, in.b7, var12, var19, 0x61);
  int var21; LUT(var21, in.b2, in.b3, var9, 0xb5);
  int var22; LUT(var22, var21, var20, in.b5, 0x97);
  int var23; LUT(var23, in.b1, var19, var22, 0xac);
  int var24; LUT(var24, in.b4, var17, var23, 0xac);
  int var25; LUT(var25, in.b3, in.b4, in.b5, 0x0d);
  int var26; LUT(var26, var25, var18, var24, 0x38);
  int var27; LUT(var27, var9, var20, var26, 0x8e);
  int var28; LUT(var28, in.b4, var10, var19, 0x1a);
  int var29; LUT(var29, var28, var27, var24, 0xa9);
  int var30; LUT(var30, in.b1, var26, var29, 0xac);
  int var31; LUT(var31, in.b7, var22, var28, 0x35);
  int var32; LUT(var32, var31, var21, var24, 0x85);
  int var33; LUT(var33, in.b5, in.b7, var19, 0xa0);
  int var34; LUT(var34, in.b3, var28, var31, 0x3d);
  int var35; LUT(var35, var34, var33, var32, 0x96);
  int var36; LUT(var36, in.b1, var32, var35, 0xac);
  int var37; LUT(var37, in.b2, var30, var36, 0xac);
  int out5;  LUT(out5, in.b6, var24, var37, 0xac);
  return out5;
}

__device__ __forceinline__ int s6(eightbits in) {
  int var8;  LUT(var8, in.b1, in.b2, in.b6, 0x61);
  int var9;  LUT(var9, in.b7, in.b6, var8, 0xac);
  int var10; LUT(var10, in.b2, in.b7, var8, 0x76);
  int var11; LUT(var11, var10, in.b1, in.b6, 0x24);
  int var12; LUT(var12, in.b5, var9, var11, 0xac);
  int var13; LUT(var13, in.b6, in.b7, var8, 0xfc);
  int var14; LUT(var14, in.b1, in.b2, in.b5, 0x5b);
  int var15; LUT(var15, var14, var13, var12, 0x59);
  int var16; LUT(var16, in.b0, var12, var15, 0xac);
  int var17; LUT(var17, var8, var15, var16, 0x73);
  int var18; LUT(var18, var17, in.b6, in.b7, 0x8b);
  int var19; LUT(var19, in.b1, in.b6, var18, 0xe8);
  int var20; LUT(var20, in.b0, in.b2, var11, 0x49);
  int var21; LUT(var21, var20, var19, var10, 0xa7);
  int var22; LUT(var22, in.b5, var18, var21, 0xac);
  int var23; LUT(var23, in.b4, var16, var22, 0xac);
  int var24; LUT(var24, in.b0, in.b5, var18, 0x5b);
  int var25; LUT(var25, var24, var8, var14, 0x29);
  int var26; LUT(var26, in.b5, var8, var13, 0x4a);
  int var27; LUT(var27, in.b0, in.b7, var12, 0x7e);
  int var28; LUT(var28, var27, var26, var18, 0x69);
  int var29; LUT(var29, in.b2, var25, var28, 0xac);
  int var30; LUT(var30, in.b6, in.b7, var8, 0x4f);
  int var31; LUT(var31, var30, in.b5, var10, 0x69);
  int var32; LUT(var32, var14, var26, var28, 0xa4);
  int var33; LUT(var33, in.b1, in.b6, var29, 0x98);
  int var34; LUT(var34, var33, var32, var13, 0x39);
  int var35; LUT(var35, in.b0, var31, var34, 0xac);
  int var36; LUT(var36, in.b4, var29, var35, 0xac);
  int out6;  LUT(out6, in.b3, var23, var36, 0xac);
  return out6;
}

__device__ __forceinline__ int s7(eightbits in) {
  int var8;  LUT(var8, in.b1, in.b0, in.b2, 0x6f);
  int var9;  LUT(var9, in.b6, var8, in.b2, 0xac);
  int var10; LUT(var10, in.b0, in.b1, in.b2, 0x5c);
  int var11; LUT(var11, var10, in.b6, var9, 0x86);
  int var12; LUT(var12, in.b3, var9, var11, 0xac);
  int var13; LUT(var13, in.b2, in.b3, var9, 0x7b);
  int var14; LUT(var14, in.b0, in.b6, var12, 0x19);
  int var15; LUT(var15, var14, var13, var8, 0xb6);
  int var16; LUT(var16, in.b4, var12, var15, 0xac);
  int var17; LUT(var17, in.b2, in.b6, var15, 0x72);
  int var18; LUT(var18, in.b0, in.b1, in.b3, 0xa9);
  int var19; LUT(var19, var18, var17, var14, 0x36);
  int var20; LUT(var20, in.b6, var10, var18, 0xd6);
  int var21; LUT(var21, in.b1, var12, var13, 0x46);
  int var22; LUT(var22, var21, var20, var17, 0xcb);
  int var23; LUT(var23, in.b4, var19, var22, 0xac);
  int var24; LUT(var24, in.b7, var16, var23, 0xac);
  int var25; LUT(var25, in.b7, var19, var20, 0x79);
  int var26; LUT(var26, in.b3, var11, var17, 0x3e);
  int var27; LUT(var27, var26, var25, var9, 0x6a);
  int var28; LUT(var28, in.b7, var9, var13, 0x4d);
  int var29; LUT(var29, var28, in.b3, var20, 0x96);
  int var30; LUT(var30, in.b4, var27, var29, 0xac);
  int var31; LUT(var31, var16, var27, var29, 0xf2);
  int var32; LUT(var32, in.b3, in.b7, var23, 0xe6);
  int var33; LUT(var33, var32, var31, var13, 0x92);
  int var34; LUT(var34, in.b7, var16, var29, 0xd4);
  int var35; LUT(var35, var34, var22, var30, 0x16);
  int var36; LUT(var36, in.b6, var33, var35, 0xac);
  int var37; LUT(var37, in.b1, var30, var36, 0xac);
  int out7;  LUT(out7, in.b5, var24, var37, 0xac);
  return out7;
}

/* Bitsliced single round encryption. Used by the brute force search algorithm
   in an unrolled loop. */
template <bool last>
__device__ __forceinline__ twentyfourbits encrypt_round(twentyfourbits bits, int round,
    volatile const int * __restrict key_hi) {

  volatile int bidx = blockIdx.x;
  volatile int tidx = threadIdx.x;

  eightbits cur;

  /* Calculate round tweak offsets. */
  int tw_off_a, tw_off_b, tw_off_c;
  switch (round % 8) {
    case 0:
      tw_off_a = 16;
      tw_off_c = 8;
      tw_off_b = 0;
      break;
    case 1:
      tw_off_a = 56;
      tw_off_c = 48;
      tw_off_b = 40;
      break;
    case 2:
      tw_off_a = 32;
      tw_off_c = 24;
      tw_off_b = 16;
      break;
    case 3:
      tw_off_a = 8;
      tw_off_c = 0;
      tw_off_b = 56;
      break;
    case 4:
      tw_off_a = 48;
      tw_off_c = 40;
      tw_off_b = 32;
      break;
    case 5:
      tw_off_a = 24;
      tw_off_c = 16;
      tw_off_b = 8;
      break;
    case 6:
      tw_off_a = 0;
      tw_off_c = 56;
      tw_off_b = 48;
      break;
    case 7:
      tw_off_a = 40;
      tw_off_c = 32;
      tw_off_b = 24;
      break;
  }

  /* A xor B xor key xor tweak. */
  switch (round % 7) {
    case 0:
      cur.b7 = (0 - ((bidx >> 8)  & 1)) ^ tw1_c[tw_off_a + 7] ^ bits.b.b7 ^ bits.a.b7;
      cur.b6 = (0 - ((bidx >> 7)  & 1)) ^ tw1_c[tw_off_a + 6] ^ bits.b.b6 ^ bits.a.b6;
      cur.b5 = (0 - ((bidx >> 6)  & 1)) ^ tw1_c[tw_off_a + 5] ^ bits.b.b5 ^ bits.a.b5;
      cur.b4 = (0 - ((bidx >> 5)  & 1)) ^ tw1_c[tw_off_a + 4] ^ bits.b.b4 ^ bits.a.b4;
      cur.b3 = (0 - ((bidx >> 4)  & 1)) ^ tw1_c[tw_off_a + 3] ^ bits.b.b3 ^ bits.a.b3;
      cur.b2 = (0 - ((bidx >> 3)  & 1)) ^ tw1_c[tw_off_a + 2] ^ bits.b.b2 ^ bits.a.b2;
      cur.b1 = (0 - ((bidx >> 2)  & 1)) ^ tw1_c[tw_off_a + 1] ^ bits.b.b1 ^ bits.a.b1;
      cur.b0 = (0 - ((bidx >> 1)  & 1)) ^ tw1_c[tw_off_a]     ^ bits.b.b0 ^ bits.a.b0;
      break;
    case 1:
      cur.b7 = key_hi[23]               ^ tw1_c[tw_off_a + 7] ^ bits.b.b7 ^ bits.a.b7;
      cur.b6 = key_hi[22]               ^ tw1_c[tw_off_a + 6] ^ bits.b.b6 ^ bits.a.b6;
      cur.b5 = key_hi[21]               ^ tw1_c[tw_off_a + 5] ^ bits.b.b5 ^ bits.a.b5;
      cur.b4 = key_hi[20]               ^ tw1_c[tw_off_a + 4] ^ bits.b.b4 ^ bits.a.b4;
      cur.b3 = key_hi[19]               ^ tw1_c[tw_off_a + 3] ^ bits.b.b3 ^ bits.a.b3;
      cur.b2 = key_hi[18]               ^ tw1_c[tw_off_a + 2] ^ bits.b.b2 ^ bits.a.b2;
      cur.b1 = key_hi[17]               ^ tw1_c[tw_off_a + 1] ^ bits.b.b1 ^ bits.a.b1;
      cur.b0 = key_hi[16]               ^ tw1_c[tw_off_a]     ^ bits.b.b0 ^ bits.a.b0;
      break;
    case 2:
      cur.b7 = (0 - ((bidx >> 16) & 1)) ^ tw1_c[tw_off_a + 7] ^ bits.b.b7 ^ bits.a.b7;
      cur.b6 = (0 - ((bidx >> 15) & 1)) ^ tw1_c[tw_off_a + 6] ^ bits.b.b6 ^ bits.a.b6;
      cur.b5 = (0 - ((bidx >> 14) & 1)) ^ tw1_c[tw_off_a + 5] ^ bits.b.b5 ^ bits.a.b5;
      cur.b4 = (0 - ((bidx >> 13) & 1)) ^ tw1_c[tw_off_a + 4] ^ bits.b.b4 ^ bits.a.b4;
      cur.b3 = (0 - ((bidx >> 12) & 1)) ^ tw1_c[tw_off_a + 3] ^ bits.b.b3 ^ bits.a.b3;
      cur.b2 = (0 - ((bidx >> 11) & 1)) ^ tw1_c[tw_off_a + 2] ^ bits.b.b2 ^ bits.a.b2;
      cur.b1 = (0 - ((bidx >> 10) & 1)) ^ tw1_c[tw_off_a + 1] ^ bits.b.b1 ^ bits.a.b1;
      cur.b0 = (0 - ((bidx >> 9)  & 1)) ^ tw1_c[tw_off_a]     ^ bits.b.b0 ^ bits.a.b0;
      break;
    case 3:
      cur.b7 = (0 - ((tidx >> 2)  & 1)) ^ tw1_c[tw_off_a + 7] ^ bits.b.b7 ^ bits.a.b7;
      cur.b6 = (0 - ((tidx >> 1)  & 1)) ^ tw1_c[tw_off_a + 6] ^ bits.b.b6 ^ bits.a.b6;
      cur.b5 = (0 -  (tidx        & 1)) ^ tw1_c[tw_off_a + 5] ^ bits.b.b5 ^ bits.a.b5;
      cur.b4 = 0xffff0000               ^ tw1_c[tw_off_a + 4] ^ bits.b.b4 ^ bits.a.b4;
      cur.b3 = 0xff00ff00               ^ tw1_c[tw_off_a + 3] ^ bits.b.b3 ^ bits.a.b3;
      cur.b2 = 0xf0f0f0f0               ^ tw1_c[tw_off_a + 2] ^ bits.b.b2 ^ bits.a.b2;
      cur.b1 = 0xcccccccc               ^ tw1_c[tw_off_a + 1] ^ bits.b.b1 ^ bits.a.b1;
      cur.b0 = 0xaaaaaaaa               ^ tw1_c[tw_off_a]     ^ bits.b.b0 ^ bits.a.b0;
      break;
    case 4:
      cur.b7 = key_hi[7]                ^ tw1_c[tw_off_a + 7] ^ bits.b.b7 ^ bits.a.b7;
      cur.b6 = key_hi[6]                ^ tw1_c[tw_off_a + 6] ^ bits.b.b6 ^ bits.a.b6;
      cur.b5 = key_hi[5]                ^ tw1_c[tw_off_a + 5] ^ bits.b.b5 ^ bits.a.b5;
      cur.b4 = key_hi[4]                ^ tw1_c[tw_off_a + 4] ^ bits.b.b4 ^ bits.a.b4;
      cur.b3 = key_hi[3]                ^ tw1_c[tw_off_a + 3] ^ bits.b.b3 ^ bits.a.b3;
      cur.b2 = key_hi[2]                ^ tw1_c[tw_off_a + 2] ^ bits.b.b2 ^ bits.a.b2;
      cur.b1 = key_hi[1]                ^ tw1_c[tw_off_a + 1] ^ bits.b.b1 ^ bits.a.b1;
      cur.b0 = key_hi[0]                ^ tw1_c[tw_off_a]     ^ bits.b.b0 ^ bits.a.b0;
      break;
    case 5:
      cur.b7 = (0 -  (bidx        & 1)) ^ tw1_c[tw_off_a + 7] ^ bits.b.b7 ^ bits.a.b7;
      cur.b6 = (0 - ((tidx >> 9)  & 1)) ^ tw1_c[tw_off_a + 6] ^ bits.b.b6 ^ bits.a.b6;
      cur.b5 = (0 - ((tidx >> 8)  & 1)) ^ tw1_c[tw_off_a + 5] ^ bits.b.b5 ^ bits.a.b5;
      cur.b4 = (0 - ((tidx >> 7)  & 1)) ^ tw1_c[tw_off_a + 4] ^ bits.b.b4 ^ bits.a.b4;
      cur.b3 = (0 - ((tidx >> 6)  & 1)) ^ tw1_c[tw_off_a + 3] ^ bits.b.b3 ^ bits.a.b3;
      cur.b2 = (0 - ((tidx >> 5)  & 1)) ^ tw1_c[tw_off_a + 2] ^ bits.b.b2 ^ bits.a.b2;
      cur.b1 = (0 - ((tidx >> 4)  & 1)) ^ tw1_c[tw_off_a + 1] ^ bits.b.b1 ^ bits.a.b1;
      cur.b0 = (0 - ((tidx >> 3)  & 1)) ^ tw1_c[tw_off_a]     ^ bits.b.b0 ^ bits.a.b0;
      break;
    case 6:
      cur.b7 = key_hi[15]               ^ tw1_c[tw_off_a + 7] ^ bits.b.b7 ^ bits.a.b7;
      cur.b6 = key_hi[14]               ^ tw1_c[tw_off_a + 6] ^ bits.b.b6 ^ bits.a.b6;
      cur.b5 = key_hi[13]               ^ tw1_c[tw_off_a + 5] ^ bits.b.b5 ^ bits.a.b5;
      cur.b4 = key_hi[12]               ^ tw1_c[tw_off_a + 4] ^ bits.b.b4 ^ bits.a.b4;
      cur.b3 = key_hi[11]               ^ tw1_c[tw_off_a + 3] ^ bits.b.b3 ^ bits.a.b3;
      cur.b2 = key_hi[10]               ^ tw1_c[tw_off_a + 2] ^ bits.b.b2 ^ bits.a.b2;
      cur.b1 = key_hi[9]                ^ tw1_c[tw_off_a + 1] ^ bits.b.b1 ^ bits.a.b1;
      cur.b0 = key_hi[8]                ^ tw1_c[tw_off_a]     ^ bits.b.b0 ^ bits.a.b0;
      break;
  }

  if (last) {
    bits.a = cur;
  } else {
    bits.a.b0 = s0(cur);
    bits.a.b1 = s1(cur);
    bits.a.b2 = s2(cur);
    bits.a.b3 = s3(cur);
    bits.a.b4 = s4(cur);
    bits.a.b5 = s5(cur);
    bits.a.b6 = s6(cur);
    bits.a.b7 = s7(cur);
  }

  /* C xor B xor key xor tweak. */
  switch (round % 7) {
    case 0:
      cur.b7 = (0 -  (bidx        & 1)) ^ tw1_c[tw_off_c + 7] ^ bits.b.b7 ^ bits.c.b7;
      cur.b6 = (0 - ((tidx >> 9)  & 1)) ^ tw1_c[tw_off_c + 6] ^ bits.b.b6 ^ bits.c.b6;
      cur.b5 = (0 - ((tidx >> 8)  & 1)) ^ tw1_c[tw_off_c + 5] ^ bits.b.b5 ^ bits.c.b5;
      cur.b4 = (0 - ((tidx >> 7)  & 1)) ^ tw1_c[tw_off_c + 4] ^ bits.b.b4 ^ bits.c.b4;
      cur.b3 = (0 - ((tidx >> 6)  & 1)) ^ tw1_c[tw_off_c + 3] ^ bits.b.b3 ^ bits.c.b3;
      cur.b2 = (0 - ((tidx >> 5)  & 1)) ^ tw1_c[tw_off_c + 2] ^ bits.b.b2 ^ bits.c.b2;
      cur.b1 = (0 - ((tidx >> 4)  & 1)) ^ tw1_c[tw_off_c + 1] ^ bits.b.b1 ^ bits.c.b1;
      cur.b0 = (0 - ((tidx >> 3)  & 1)) ^ tw1_c[tw_off_c]     ^ bits.b.b0 ^ bits.c.b0;
      break;
    case 1:
      cur.b7 = key_hi[15]               ^ tw1_c[tw_off_c + 7] ^ bits.b.b7 ^ bits.c.b7;
      cur.b6 = key_hi[14]               ^ tw1_c[tw_off_c + 6] ^ bits.b.b6 ^ bits.c.b6;
      cur.b5 = key_hi[13]               ^ tw1_c[tw_off_c + 5] ^ bits.b.b5 ^ bits.c.b5;
      cur.b4 = key_hi[12]               ^ tw1_c[tw_off_c + 4] ^ bits.b.b4 ^ bits.c.b4;
      cur.b3 = key_hi[11]               ^ tw1_c[tw_off_c + 3] ^ bits.b.b3 ^ bits.c.b3;
      cur.b2 = key_hi[10]               ^ tw1_c[tw_off_c + 2] ^ bits.b.b2 ^ bits.c.b2;
      cur.b1 = key_hi[9]                ^ tw1_c[tw_off_c + 1] ^ bits.b.b1 ^ bits.c.b1;
      cur.b0 = key_hi[8]                ^ tw1_c[tw_off_c]     ^ bits.b.b0 ^ bits.c.b0;
      break;
    case 2:
      cur.b7 = (0 - ((bidx >> 8)  & 1)) ^ tw1_c[tw_off_c + 7] ^ bits.b.b7 ^ bits.c.b7;
      cur.b6 = (0 - ((bidx >> 7)  & 1)) ^ tw1_c[tw_off_c + 6] ^ bits.b.b6 ^ bits.c.b6;
      cur.b5 = (0 - ((bidx >> 6)  & 1)) ^ tw1_c[tw_off_c + 5] ^ bits.b.b5 ^ bits.c.b5;
      cur.b4 = (0 - ((bidx >> 5)  & 1)) ^ tw1_c[tw_off_c + 4] ^ bits.b.b4 ^ bits.c.b4;
      cur.b3 = (0 - ((bidx >> 4)  & 1)) ^ tw1_c[tw_off_c + 3] ^ bits.b.b3 ^ bits.c.b3;
      cur.b2 = (0 - ((bidx >> 3)  & 1)) ^ tw1_c[tw_off_c + 2] ^ bits.b.b2 ^ bits.c.b2;
      cur.b1 = (0 - ((bidx >> 2)  & 1)) ^ tw1_c[tw_off_c + 1] ^ bits.b.b1 ^ bits.c.b1;
      cur.b0 = (0 - ((bidx >> 1)  & 1)) ^ tw1_c[tw_off_c]     ^ bits.b.b0 ^ bits.c.b0;
      break;
    case 3:
      cur.b7 = key_hi[23]               ^ tw1_c[tw_off_c + 7] ^ bits.b.b7 ^ bits.c.b7;
      cur.b6 = key_hi[22]               ^ tw1_c[tw_off_c + 6] ^ bits.b.b6 ^ bits.c.b6;
      cur.b5 = key_hi[21]               ^ tw1_c[tw_off_c + 5] ^ bits.b.b5 ^ bits.c.b5;
      cur.b4 = key_hi[20]               ^ tw1_c[tw_off_c + 4] ^ bits.b.b4 ^ bits.c.b4;
      cur.b3 = key_hi[19]               ^ tw1_c[tw_off_c + 3] ^ bits.b.b3 ^ bits.c.b3;
      cur.b2 = key_hi[18]               ^ tw1_c[tw_off_c + 2] ^ bits.b.b2 ^ bits.c.b2;
      cur.b1 = key_hi[17]               ^ tw1_c[tw_off_c + 1] ^ bits.b.b1 ^ bits.c.b1;
      cur.b0 = key_hi[16]               ^ tw1_c[tw_off_c]     ^ bits.b.b0 ^ bits.c.b0;
      break;
    case 4:
      cur.b7 = (0 - ((bidx >> 16) & 1)) ^ tw1_c[tw_off_c + 7] ^ bits.b.b7 ^ bits.c.b7;
      cur.b6 = (0 - ((bidx >> 15) & 1)) ^ tw1_c[tw_off_c + 6] ^ bits.b.b6 ^ bits.c.b6;
      cur.b5 = (0 - ((bidx >> 14) & 1)) ^ tw1_c[tw_off_c + 5] ^ bits.b.b5 ^ bits.c.b5;
      cur.b4 = (0 - ((bidx >> 13) & 1)) ^ tw1_c[tw_off_c + 4] ^ bits.b.b4 ^ bits.c.b4;
      cur.b3 = (0 - ((bidx >> 12) & 1)) ^ tw1_c[tw_off_c + 3] ^ bits.b.b3 ^ bits.c.b3;
      cur.b2 = (0 - ((bidx >> 11) & 1)) ^ tw1_c[tw_off_c + 2] ^ bits.b.b2 ^ bits.c.b2;
      cur.b1 = (0 - ((bidx >> 10) & 1)) ^ tw1_c[tw_off_c + 1] ^ bits.b.b1 ^ bits.c.b1;
      cur.b0 = (0 - ((bidx >> 9)  & 1)) ^ tw1_c[tw_off_c]     ^ bits.b.b0 ^ bits.c.b0;
      break;
    case 5:
      cur.b7 = (0 - ((tidx >> 2)  & 1)) ^ tw1_c[tw_off_c + 7] ^ bits.b.b7 ^ bits.c.b7;
      cur.b6 = (0 - ((tidx >> 1)  & 1)) ^ tw1_c[tw_off_c + 6] ^ bits.b.b6 ^ bits.c.b6;
      cur.b5 = (0 -  (tidx        & 1)) ^ tw1_c[tw_off_c + 5] ^ bits.b.b5 ^ bits.c.b5;
      cur.b4 = 0xffff0000               ^ tw1_c[tw_off_c + 4] ^ bits.b.b4 ^ bits.c.b4;
      cur.b3 = 0xff00ff00               ^ tw1_c[tw_off_c + 3] ^ bits.b.b3 ^ bits.c.b3;
      cur.b2 = 0xf0f0f0f0               ^ tw1_c[tw_off_c + 2] ^ bits.b.b2 ^ bits.c.b2;
      cur.b1 = 0xcccccccc               ^ tw1_c[tw_off_c + 1] ^ bits.b.b1 ^ bits.c.b1;
      cur.b0 = 0xaaaaaaaa               ^ tw1_c[tw_off_c]     ^ bits.b.b0 ^ bits.c.b0;
      break;
    case 6:
      cur.b7 = key_hi[7]                ^ tw1_c[tw_off_c + 7] ^ bits.b.b7 ^ bits.c.b7;
      cur.b6 = key_hi[6]                ^ tw1_c[tw_off_c + 6] ^ bits.b.b6 ^ bits.c.b6;
      cur.b5 = key_hi[5]                ^ tw1_c[tw_off_c + 5] ^ bits.b.b5 ^ bits.c.b5;
      cur.b4 = key_hi[4]                ^ tw1_c[tw_off_c + 4] ^ bits.b.b4 ^ bits.c.b4;
      cur.b3 = key_hi[3]                ^ tw1_c[tw_off_c + 3] ^ bits.b.b3 ^ bits.c.b3;
      cur.b2 = key_hi[2]                ^ tw1_c[tw_off_c + 2] ^ bits.b.b2 ^ bits.c.b2;
      cur.b1 = key_hi[1]                ^ tw1_c[tw_off_c + 1] ^ bits.b.b1 ^ bits.c.b1;
      cur.b0 = key_hi[0]                ^ tw1_c[tw_off_c]     ^ bits.b.b0 ^ bits.c.b0;
      break;
  }

  if (last) {
    bits.c = cur;
  } else {
    bits.c.b0 = s0(cur);
    bits.c.b1 = s1(cur);
    bits.c.b2 = s2(cur);
    bits.c.b3 = s3(cur);
    bits.c.b4 = s4(cur);
    bits.c.b5 = s5(cur);
    bits.c.b6 = s6(cur);
    bits.c.b7 = s7(cur);
  }

  /* B xor key xor tweak. */
  switch (round % 7) {
    case 0:
      cur.b7 = (0 - ((tidx >> 2) & 1))  ^ tw1_c[tw_off_b + 7] ^ bits.b.b7;
      cur.b6 = (0 - ((tidx >> 1) & 1))  ^ tw1_c[tw_off_b + 6] ^ bits.b.b6;
      cur.b5 = (0 -  (tidx       & 1))  ^ tw1_c[tw_off_b + 5] ^ bits.b.b5;
      cur.b4 = 0xffff0000               ^ tw1_c[tw_off_b + 4] ^ bits.b.b4;
      cur.b3 = 0xff00ff00               ^ tw1_c[tw_off_b + 3] ^ bits.b.b3;
      cur.b2 = 0xf0f0f0f0               ^ tw1_c[tw_off_b + 2] ^ bits.b.b2;
      cur.b1 = 0xcccccccc               ^ tw1_c[tw_off_b + 1] ^ bits.b.b1;
      cur.b0 = 0xaaaaaaaa               ^ tw1_c[tw_off_b]     ^ bits.b.b0;
      break;
    case 1:
      cur.b7 = key_hi[7]                ^ tw1_c[tw_off_b + 7] ^ bits.b.b7;
      cur.b6 = key_hi[6]                ^ tw1_c[tw_off_b + 6] ^ bits.b.b6;
      cur.b5 = key_hi[5]                ^ tw1_c[tw_off_b + 5] ^ bits.b.b5;
      cur.b4 = key_hi[4]                ^ tw1_c[tw_off_b + 4] ^ bits.b.b4;
      cur.b3 = key_hi[3]                ^ tw1_c[tw_off_b + 3] ^ bits.b.b3;
      cur.b2 = key_hi[2]                ^ tw1_c[tw_off_b + 2] ^ bits.b.b2;
      cur.b1 = key_hi[1]                ^ tw1_c[tw_off_b + 1] ^ bits.b.b1;
      cur.b0 = key_hi[0]                ^ tw1_c[tw_off_b]     ^ bits.b.b0;
      break;
    case 2:
      cur.b7 = (0 -  (bidx        & 1)) ^ tw1_c[tw_off_b + 7] ^ bits.b.b7;
      cur.b6 = (0 - ((tidx >> 9)  & 1)) ^ tw1_c[tw_off_b + 6] ^ bits.b.b6;
      cur.b5 = (0 - ((tidx >> 8)  & 1)) ^ tw1_c[tw_off_b + 5] ^ bits.b.b5;
      cur.b4 = (0 - ((tidx >> 7)  & 1)) ^ tw1_c[tw_off_b + 4] ^ bits.b.b4;
      cur.b3 = (0 - ((tidx >> 6)  & 1)) ^ tw1_c[tw_off_b + 3] ^ bits.b.b3;
      cur.b2 = (0 - ((tidx >> 5)  & 1)) ^ tw1_c[tw_off_b + 2] ^ bits.b.b2;
      cur.b1 = (0 - ((tidx >> 4)  & 1)) ^ tw1_c[tw_off_b + 1] ^ bits.b.b1;
      cur.b0 = (0 - ((tidx >> 3)  & 1)) ^ tw1_c[tw_off_b]     ^ bits.b.b0;
      break;
    case 3:
      cur.b7 = key_hi[15]               ^ tw1_c[tw_off_b + 7] ^ bits.b.b7;
      cur.b6 = key_hi[14]               ^ tw1_c[tw_off_b + 6] ^ bits.b.b6;
      cur.b5 = key_hi[13]               ^ tw1_c[tw_off_b + 5] ^ bits.b.b5;
      cur.b4 = key_hi[12]               ^ tw1_c[tw_off_b + 4] ^ bits.b.b4;
      cur.b3 = key_hi[11]               ^ tw1_c[tw_off_b + 3] ^ bits.b.b3;
      cur.b2 = key_hi[10]               ^ tw1_c[tw_off_b + 2] ^ bits.b.b2;
      cur.b1 = key_hi[9]                ^ tw1_c[tw_off_b + 1] ^ bits.b.b1;
      cur.b0 = key_hi[8]                ^ tw1_c[tw_off_b]     ^ bits.b.b0;
      break;
    case 4:
      cur.b7 = (0 - ((bidx >> 8)  & 1)) ^ tw1_c[tw_off_b + 7] ^ bits.b.b7;
      cur.b6 = (0 - ((bidx >> 7)  & 1)) ^ tw1_c[tw_off_b + 6] ^ bits.b.b6;
      cur.b5 = (0 - ((bidx >> 6)  & 1)) ^ tw1_c[tw_off_b + 5] ^ bits.b.b5;
      cur.b4 = (0 - ((bidx >> 5)  & 1)) ^ tw1_c[tw_off_b + 4] ^ bits.b.b4;
      cur.b3 = (0 - ((bidx >> 4)  & 1)) ^ tw1_c[tw_off_b + 3] ^ bits.b.b3;
      cur.b2 = (0 - ((bidx >> 3)  & 1)) ^ tw1_c[tw_off_b + 2] ^ bits.b.b2;
      cur.b1 = (0 - ((bidx >> 2)  & 1)) ^ tw1_c[tw_off_b + 1] ^ bits.b.b1;
      cur.b0 = (0 - ((bidx >> 1)  & 1)) ^ tw1_c[tw_off_b]     ^ bits.b.b0;
      break;
    case 5:
      cur.b7 = key_hi[23]               ^ tw1_c[tw_off_b + 7] ^ bits.b.b7;
      cur.b6 = key_hi[22]               ^ tw1_c[tw_off_b + 6] ^ bits.b.b6;
      cur.b5 = key_hi[21]               ^ tw1_c[tw_off_b + 5] ^ bits.b.b5;
      cur.b4 = key_hi[20]               ^ tw1_c[tw_off_b + 4] ^ bits.b.b4;
      cur.b3 = key_hi[19]               ^ tw1_c[tw_off_b + 3] ^ bits.b.b3;
      cur.b2 = key_hi[18]               ^ tw1_c[tw_off_b + 2] ^ bits.b.b2;
      cur.b1 = key_hi[17]               ^ tw1_c[tw_off_b + 1] ^ bits.b.b1;
      cur.b0 = key_hi[16]               ^ tw1_c[tw_off_b]     ^ bits.b.b0;
      break;
    case 6:
      cur.b7 = (0 - ((bidx >> 16) & 1)) ^ tw1_c[tw_off_b + 7] ^ bits.b.b7;
      cur.b6 = (0 - ((bidx >> 15) & 1)) ^ tw1_c[tw_off_b + 6] ^ bits.b.b6;
      cur.b5 = (0 - ((bidx >> 14) & 1)) ^ tw1_c[tw_off_b + 5] ^ bits.b.b5;
      cur.b4 = (0 - ((bidx >> 13) & 1)) ^ tw1_c[tw_off_b + 4] ^ bits.b.b4;
      cur.b3 = (0 - ((bidx >> 12) & 1)) ^ tw1_c[tw_off_b + 3] ^ bits.b.b3;
      cur.b2 = (0 - ((bidx >> 11) & 1)) ^ tw1_c[tw_off_b + 2] ^ bits.b.b2;
      cur.b1 = (0 - ((bidx >> 10) & 1)) ^ tw1_c[tw_off_b + 1] ^ bits.b.b1;
      cur.b0 = (0 - ((bidx >> 9)  & 1)) ^ tw1_c[tw_off_b]     ^ bits.b.b0;
      break;
  }

  if (last) {
    bits.b = cur;
  } else {
    cur.b0 ^= bits.a.b0 ^ bits.c.b0;
    cur.b1 ^= bits.a.b1 ^ bits.c.b1;
    cur.b2 ^= bits.a.b2 ^ bits.c.b2;
    cur.b3 ^= bits.a.b3 ^ bits.c.b3;
    cur.b4 ^= bits.a.b4 ^ bits.c.b4;
    cur.b5 ^= bits.a.b5 ^ bits.c.b5;
    cur.b6 ^= bits.a.b6 ^ bits.c.b6;
    cur.b7 ^= bits.a.b7 ^ bits.c.b7;
    bits.b.b0 = s0(cur);
    bits.b.b1 = s1(cur);
    bits.b.b2 = s2(cur);
    bits.b.b3 = s3(cur);
    bits.b.b4 = s4(cur);
    bits.b.b5 = s5(cur);
    bits.b.b6 = s6(cur);
    bits.b.b7 = s7(cur);
  }

  return bits;
}

/* Brute force search for key. The five least significant bytes of matching
   keys are placed in ret. */
template<int rounds>
__launch_bounds__(1024, 1)
__global__ void brute_force(int *ret, volatile int off) {
  __shared__ int pt[24];
  __shared__ int ct[24];
  __shared__ int key_12[24];
  __shared__ int found[100];
  if (threadIdx.x < 24) {
    pt[threadIdx.x] = pt1_c[threadIdx.x + off];
    ct[threadIdx.x] = ct1_c[threadIdx.x + off];
    key_12[threadIdx.x] = key_c[threadIdx.x + off];
    found[threadIdx.x] = 0;
  }

  __syncthreads();

  twentyfourbits bits;
  bits.a.b7 = pt[23];
  bits.a.b6 = pt[22];
  bits.a.b5 = pt[21];
  bits.a.b4 = pt[20];
  bits.a.b3 = pt[19];
  bits.a.b2 = pt[18];
  bits.a.b1 = pt[17];
  bits.a.b0 = pt[16];
  bits.b.b7 = pt[15];
  bits.b.b6 = pt[14];
  bits.b.b5 = pt[13];
  bits.b.b4 = pt[12];
  bits.b.b3 = pt[11];
  bits.b.b2 = pt[10];
  bits.b.b1 = pt[9];
  bits.b.b0 = pt[8];
  bits.c.b7 = pt[7];
  bits.c.b6 = pt[6];
  bits.c.b5 = pt[5];
  bits.c.b4 = pt[4];
  bits.c.b3 = pt[3];
  bits.c.b2 = pt[2];
  bits.c.b1 = pt[1];
  bits.c.b0 = pt[0];

  #pragma unroll
  for (int i = 2; i < rounds; i++) {
    bits = encrypt_round<false>(bits, i, &key_12[0]);
  }
  bits = encrypt_round<true>(bits, rounds, &key_12[0]);

  int rr = 0;
  rr |= bits.a.b7 ^ ct[23];
  rr |= bits.a.b6 ^ ct[22];
  rr |= bits.a.b5 ^ ct[21];
  rr |= bits.a.b4 ^ ct[20];
  rr |= bits.a.b3 ^ ct[19];
  rr |= bits.a.b2 ^ ct[18];
  rr |= bits.a.b1 ^ ct[17];
  rr |= bits.a.b0 ^ ct[16];
  rr |= bits.b.b7 ^ ct[15];
  rr |= bits.b.b6 ^ ct[14];
  rr |= bits.b.b5 ^ ct[13];
  rr |= bits.b.b4 ^ ct[12];
  rr |= bits.b.b3 ^ ct[11];
  rr |= bits.b.b2 ^ ct[10];
  rr |= bits.b.b1 ^ ct[9];
  rr |= bits.b.b0 ^ ct[8];
  rr |= bits.c.b7 ^ ct[7];
  rr |= bits.c.b6 ^ ct[6];
  rr |= bits.c.b5 ^ ct[5];
  rr |= bits.c.b4 ^ ct[4];
  rr |= bits.c.b3 ^ ct[3];
  rr |= bits.c.b2 ^ ct[2];
  rr |= bits.c.b1 ^ ct[1];
  rr |= bits.c.b0 ^ ct[0];

  /* Put matches in shared memory. */
  int ptr;
  if (rr != 0xffffffff) {
    ptr = atomicAdd_block(found, 1);
    found[ptr + 2] = (blockIdx.x << 10) | threadIdx.x;
    found[ptr + 3] = rr;
  }

  __syncthreads();

  if (found[0] == 0) {
    return;
  }

  /* Get global memory offset for matches found in block. */
  if (threadIdx.x == 0) {
    found[1] = atomicAdd(ret, found[0]) * 2 + 1;
  }

  __syncthreads();

  /* Copy matches to global memory. */
  if (threadIdx.x < found[0]) {
    ptr = found[1] + threadIdx.x * 2;
    ret[ptr]     = found[(threadIdx.x * 2) + 2];
    ret[ptr + 1] = found[(threadIdx.x * 2) + 3];
  }
}

__device__ __forceinline__ eightbits sbox(eightbits in) {
  eightbits out;
  out.b0 = s0(in);
  out.b1 = s1(in);
  out.b2 = s2(in);
  out.b3 = s3(in);
  out.b4 = s4(in);
  out.b5 = s5(in);
  out.b6 = s6(in);
  out.b7 = s7(in);
  return out;
}

/* Tests candidates found by find_candidates. Used when cracking 6, 7, and 8
   rounds. The five least significant bytes of matching keys are placed in ret. */
template <int rounds>
__launch_bounds__(1024, 1)
__global__ void test_candidates(int *in, int *out, int num_candidates, int offset) {

  int ptr = ((blockIdx.x << 7) | (threadIdx.x >> 3));
  if (ptr >= num_candidates) {
    return;
  }

  volatile __shared__ int pt1[24];
  volatile __shared__ int ct1[24];
  volatile __shared__ int tw1[64];
  volatile __shared__ int key_12[16];
  volatile __shared__ int k3456[1024];
  __shared__ int found[1024];
  __shared__ eightbits aa[1024];

  if (threadIdx.x < 16) {
    key_12[threadIdx.x] = key_c[threadIdx.x + offset * 16];
  }
  if (threadIdx.x < 24) {
    pt1[threadIdx.x] = pt1_c[threadIdx.x + offset * 24];
    ct1[threadIdx.x] = ct1_c[threadIdx.x + offset * 24];
  }
  if (threadIdx.x < 64) {
    tw1[threadIdx.x] = tw1_c[threadIdx.x + offset * 64];
  }

  __syncthreads();

  k3456[threadIdx.x] = in[ptr + 1];

  eightbits bb, cc;

  /* Round 1. */
  bb.b0 = pt1[8];
  bb.b1 = pt1[9];
  bb.b2 = pt1[10];
  bb.b3 = pt1[11];
  bb.b4 = pt1[12];
  bb.b5 = pt1[13];
  bb.b6 = pt1[14];
  bb.b7 = pt1[15];
  aa[threadIdx.x].b0 = pt1[16] ^ bb.b0 ^ key_12[8]  ^ tw1[56];
  aa[threadIdx.x].b1 = pt1[17] ^ bb.b1 ^ key_12[9]  ^ tw1[57];
  aa[threadIdx.x].b2 = pt1[18] ^ bb.b2 ^ key_12[10] ^ tw1[58];
  aa[threadIdx.x].b3 = pt1[19] ^ bb.b3 ^ key_12[11] ^ tw1[59];
  aa[threadIdx.x].b4 = pt1[20] ^ bb.b4 ^ key_12[12] ^ tw1[60];
  aa[threadIdx.x].b5 = pt1[21] ^ bb.b5 ^ key_12[13] ^ tw1[61];
  aa[threadIdx.x].b6 = pt1[22] ^ bb.b6 ^ key_12[14] ^ tw1[62];
  aa[threadIdx.x].b7 = pt1[23] ^ bb.b7 ^ key_12[15] ^ tw1[63];
  aa[threadIdx.x] = sbox(aa[threadIdx.x]);

  cc.b0 = pt1[0]  ^ bb.b0 ^ key_12[0] ^ tw1[48];
  cc.b1 = pt1[1]  ^ bb.b1 ^ key_12[1] ^ tw1[49];
  cc.b2 = pt1[2]  ^ bb.b2 ^ key_12[2] ^ tw1[50];
  cc.b3 = pt1[3]  ^ bb.b3 ^ key_12[3] ^ tw1[51];
  cc.b4 = pt1[4]  ^ bb.b4 ^ key_12[4] ^ tw1[52];
  cc.b5 = pt1[5]  ^ bb.b5 ^ key_12[5] ^ tw1[53];
  cc.b6 = pt1[6]  ^ bb.b6 ^ key_12[6] ^ tw1[54];
  cc.b7 = pt1[7]  ^ bb.b7 ^ key_12[7] ^ tw1[55];
  cc = sbox(cc);

  bb.b0 ^= aa[threadIdx.x].b0  ^ cc.b0 ^ (0 - ((k3456[threadIdx.x] >> 24) & 1)) ^ tw1[40];
  bb.b1 ^= aa[threadIdx.x].b1  ^ cc.b1 ^ (0 - ((k3456[threadIdx.x] >> 25) & 1)) ^ tw1[41];
  bb.b2 ^= aa[threadIdx.x].b2  ^ cc.b2 ^ (0 - ((k3456[threadIdx.x] >> 26) & 1)) ^ tw1[42];
  bb.b3 ^= aa[threadIdx.x].b3  ^ cc.b3 ^ (0 - ((k3456[threadIdx.x] >> 27) & 1)) ^ tw1[43];
  bb.b4 ^= aa[threadIdx.x].b4  ^ cc.b4 ^ (0 - ((k3456[threadIdx.x] >> 28) & 1)) ^ tw1[44];
  bb.b5 ^= aa[threadIdx.x].b5  ^ cc.b5 ^ (0 - ((k3456[threadIdx.x] >> 29) & 1)) ^ tw1[45];
  bb.b6 ^= aa[threadIdx.x].b6  ^ cc.b6 ^ (0 - ((k3456[threadIdx.x] >> 30) & 1)) ^ tw1[46];
  bb.b7 ^= aa[threadIdx.x].b7  ^ cc.b7 ^ (0 - ((k3456[threadIdx.x] >> 31) & 1)) ^ tw1[47];
  bb = sbox(bb);

  /* Round 2. */
  aa[threadIdx.x].b0 ^= bb.b0 ^ (0 - ((k3456[threadIdx.x] >> 16) & 1)) ^ tw1[32];
  aa[threadIdx.x].b1 ^= bb.b1 ^ (0 - ((k3456[threadIdx.x] >> 17) & 1)) ^ tw1[33];
  aa[threadIdx.x].b2 ^= bb.b2 ^ (0 - ((k3456[threadIdx.x] >> 18) & 1)) ^ tw1[34];
  aa[threadIdx.x].b3 ^= bb.b3 ^ (0 - ((k3456[threadIdx.x] >> 19) & 1)) ^ tw1[35];
  aa[threadIdx.x].b4 ^= bb.b4 ^ (0 - ((k3456[threadIdx.x] >> 20) & 1)) ^ tw1[36];
  aa[threadIdx.x].b5 ^= bb.b5 ^ (0 - ((k3456[threadIdx.x] >> 21) & 1)) ^ tw1[37];
  aa[threadIdx.x].b6 ^= bb.b6 ^ (0 - ((k3456[threadIdx.x] >> 22) & 1)) ^ tw1[38];
  aa[threadIdx.x].b7 ^= bb.b7 ^ (0 - ((k3456[threadIdx.x] >> 23) & 1)) ^ tw1[39];
  aa[threadIdx.x] = sbox(aa[threadIdx.x]);

  cc.b0 ^= bb.b0 ^ (0 - ((k3456[threadIdx.x] >> 8)  & 1)) ^ tw1[24];
  cc.b1 ^= bb.b1 ^ (0 - ((k3456[threadIdx.x] >> 9)  & 1)) ^ tw1[25];
  cc.b2 ^= bb.b2 ^ (0 - ((k3456[threadIdx.x] >> 10) & 1)) ^ tw1[26];
  cc.b3 ^= bb.b3 ^ (0 - ((k3456[threadIdx.x] >> 11) & 1)) ^ tw1[27];
  cc.b4 ^= bb.b4 ^ (0 - ((k3456[threadIdx.x] >> 12) & 1)) ^ tw1[28];
  cc.b5 ^= bb.b5 ^ (0 - ((k3456[threadIdx.x] >> 13) & 1)) ^ tw1[29];
  cc.b6 ^= bb.b6 ^ (0 - ((k3456[threadIdx.x] >> 14) & 1)) ^ tw1[30];
  cc.b7 ^= bb.b7 ^ (0 - ((k3456[threadIdx.x] >> 15) & 1)) ^ tw1[31];
  cc = sbox(cc);

  bb.b0 ^= aa[threadIdx.x].b0 ^ cc.b0 ^ (0 - ((k3456[threadIdx.x] >> 0) & 1)) ^ tw1[16];
  bb.b1 ^= aa[threadIdx.x].b1 ^ cc.b1 ^ (0 - ((k3456[threadIdx.x] >> 1) & 1)) ^ tw1[17];
  bb.b2 ^= aa[threadIdx.x].b2 ^ cc.b2 ^ (0 - ((k3456[threadIdx.x] >> 2) & 1)) ^ tw1[18];
  bb.b3 ^= aa[threadIdx.x].b3 ^ cc.b3 ^ (0 - ((k3456[threadIdx.x] >> 3) & 1)) ^ tw1[19];
  bb.b4 ^= aa[threadIdx.x].b4 ^ cc.b4 ^ (0 - ((k3456[threadIdx.x] >> 4) & 1)) ^ tw1[20];
  bb.b5 ^= aa[threadIdx.x].b5 ^ cc.b5 ^ (0 - ((k3456[threadIdx.x] >> 5) & 1)) ^ tw1[21];
  bb.b6 ^= aa[threadIdx.x].b6 ^ cc.b6 ^ (0 - ((k3456[threadIdx.x] >> 6) & 1)) ^ tw1[22];
  bb.b7 ^= aa[threadIdx.x].b7 ^ cc.b7 ^ (0 - ((k3456[threadIdx.x] >> 7) & 1)) ^ tw1[23];
  bb = sbox(bb);

  /* Round 3. */
  aa[threadIdx.x].b0 ^= bb.b0 ^ 0xaaaaaaaa                     ^ tw1[8];
  aa[threadIdx.x].b1 ^= bb.b1 ^ 0xcccccccc                     ^ tw1[9];
  aa[threadIdx.x].b2 ^= bb.b2 ^ 0xf0f0f0f0                     ^ tw1[10];
  aa[threadIdx.x].b3 ^= bb.b3 ^ 0xff00ff00                     ^ tw1[11];
  aa[threadIdx.x].b4 ^= bb.b4 ^ 0xffff0000                     ^ tw1[12];
  aa[threadIdx.x].b5 ^= bb.b5 ^ (0 -  (threadIdx.x       & 1)) ^ tw1[13];
  aa[threadIdx.x].b6 ^= bb.b6 ^ (0 - ((threadIdx.x >> 1) & 1)) ^ tw1[14];
  aa[threadIdx.x].b7 ^= bb.b7 ^ (0 - ((threadIdx.x >> 2) & 1)) ^ tw1[15];
  aa[threadIdx.x] = sbox(aa[threadIdx.x]);

  cc.b0 ^= bb.b0 ^ key_12[8] ^  tw1[0];
  cc.b1 ^= bb.b1 ^ key_12[9] ^  tw1[1];
  cc.b2 ^= bb.b2 ^ key_12[10] ^ tw1[2];
  cc.b3 ^= bb.b3 ^ key_12[11] ^ tw1[3];
  cc.b4 ^= bb.b4 ^ key_12[12] ^ tw1[4];
  cc.b5 ^= bb.b5 ^ key_12[13] ^ tw1[5];
  cc.b6 ^= bb.b6 ^ key_12[14] ^ tw1[6];
  cc.b7 ^= bb.b7 ^ key_12[15] ^ tw1[7];
  cc = sbox(cc);

  bb.b0 ^= aa[threadIdx.x].b0 ^ cc.b0 ^ key_12[0] ^ tw1[56];
  bb.b1 ^= aa[threadIdx.x].b1 ^ cc.b1 ^ key_12[1] ^ tw1[57];
  bb.b2 ^= aa[threadIdx.x].b2 ^ cc.b2 ^ key_12[2] ^ tw1[58];
  bb.b3 ^= aa[threadIdx.x].b3 ^ cc.b3 ^ key_12[3] ^ tw1[59];
  bb.b4 ^= aa[threadIdx.x].b4 ^ cc.b4 ^ key_12[4] ^ tw1[60];
  bb.b5 ^= aa[threadIdx.x].b5 ^ cc.b5 ^ key_12[5] ^ tw1[61];
  bb.b6 ^= aa[threadIdx.x].b6 ^ cc.b6 ^ key_12[6] ^ tw1[62];
  bb.b7 ^= aa[threadIdx.x].b7 ^ cc.b7 ^ key_12[7] ^ tw1[63];
  bb = sbox(bb);

  /* Round 4. */
  aa[threadIdx.x].b0 ^= bb.b0 ^ (0 - ((k3456[threadIdx.x] >> 24) & 1)) ^ tw1[48];
  aa[threadIdx.x].b1 ^= bb.b1 ^ (0 - ((k3456[threadIdx.x] >> 25) & 1)) ^ tw1[49];
  aa[threadIdx.x].b2 ^= bb.b2 ^ (0 - ((k3456[threadIdx.x] >> 26) & 1)) ^ tw1[50];
  aa[threadIdx.x].b3 ^= bb.b3 ^ (0 - ((k3456[threadIdx.x] >> 27) & 1)) ^ tw1[51];
  aa[threadIdx.x].b4 ^= bb.b4 ^ (0 - ((k3456[threadIdx.x] >> 28) & 1)) ^ tw1[52];
  aa[threadIdx.x].b5 ^= bb.b5 ^ (0 - ((k3456[threadIdx.x] >> 29) & 1)) ^ tw1[53];
  aa[threadIdx.x].b6 ^= bb.b6 ^ (0 - ((k3456[threadIdx.x] >> 30) & 1)) ^ tw1[54];
  aa[threadIdx.x].b7 ^= bb.b7 ^ (0 - ((k3456[threadIdx.x] >> 31) & 1)) ^ tw1[55];
  aa[threadIdx.x] = sbox(aa[threadIdx.x]);

  cc.b0 ^= bb.b0 ^ (0 - ((k3456[threadIdx.x] >> 16) & 1)) ^ tw1[40];
  cc.b1 ^= bb.b1 ^ (0 - ((k3456[threadIdx.x] >> 17) & 1)) ^ tw1[41];
  cc.b2 ^= bb.b2 ^ (0 - ((k3456[threadIdx.x] >> 18) & 1)) ^ tw1[42];
  cc.b3 ^= bb.b3 ^ (0 - ((k3456[threadIdx.x] >> 19) & 1)) ^ tw1[43];
  cc.b4 ^= bb.b4 ^ (0 - ((k3456[threadIdx.x] >> 20) & 1)) ^ tw1[44];
  cc.b5 ^= bb.b5 ^ (0 - ((k3456[threadIdx.x] >> 21) & 1)) ^ tw1[45];
  cc.b6 ^= bb.b6 ^ (0 - ((k3456[threadIdx.x] >> 22) & 1)) ^ tw1[46];
  cc.b7 ^= bb.b7 ^ (0 - ((k3456[threadIdx.x] >> 23) & 1)) ^ tw1[47];
  cc = sbox(cc);

  bb.b0 ^= aa[threadIdx.x].b0 ^ cc.b0 ^ (0 - ((k3456[threadIdx.x] >> 8)  & 1)) ^ tw1[32];
  bb.b1 ^= aa[threadIdx.x].b1 ^ cc.b1 ^ (0 - ((k3456[threadIdx.x] >> 9)  & 1)) ^ tw1[33];
  bb.b2 ^= aa[threadIdx.x].b2 ^ cc.b2 ^ (0 - ((k3456[threadIdx.x] >> 10) & 1)) ^ tw1[34];
  bb.b3 ^= aa[threadIdx.x].b3 ^ cc.b3 ^ (0 - ((k3456[threadIdx.x] >> 11) & 1)) ^ tw1[35];
  bb.b4 ^= aa[threadIdx.x].b4 ^ cc.b4 ^ (0 - ((k3456[threadIdx.x] >> 12) & 1)) ^ tw1[36];
  bb.b5 ^= aa[threadIdx.x].b5 ^ cc.b5 ^ (0 - ((k3456[threadIdx.x] >> 13) & 1)) ^ tw1[37];
  bb.b6 ^= aa[threadIdx.x].b6 ^ cc.b6 ^ (0 - ((k3456[threadIdx.x] >> 14) & 1)) ^ tw1[38];
  bb.b7 ^= aa[threadIdx.x].b7 ^ cc.b7 ^ (0 - ((k3456[threadIdx.x] >> 15) & 1)) ^ tw1[39];
  bb = sbox(bb);

  /* Round 5. */
  aa[threadIdx.x].b0 ^= bb.b0 ^ (0 - ((k3456[threadIdx.x] >> 0) & 1)) ^ tw1[24];
  aa[threadIdx.x].b1 ^= bb.b1 ^ (0 - ((k3456[threadIdx.x] >> 1) & 1)) ^ tw1[25];
  aa[threadIdx.x].b2 ^= bb.b2 ^ (0 - ((k3456[threadIdx.x] >> 2) & 1)) ^ tw1[26];
  aa[threadIdx.x].b3 ^= bb.b3 ^ (0 - ((k3456[threadIdx.x] >> 3) & 1)) ^ tw1[27];
  aa[threadIdx.x].b4 ^= bb.b4 ^ (0 - ((k3456[threadIdx.x] >> 4) & 1)) ^ tw1[28];
  aa[threadIdx.x].b5 ^= bb.b5 ^ (0 - ((k3456[threadIdx.x] >> 5) & 1)) ^ tw1[29];
  aa[threadIdx.x].b6 ^= bb.b6 ^ (0 - ((k3456[threadIdx.x] >> 6) & 1)) ^ tw1[30];
  aa[threadIdx.x].b7 ^= bb.b7 ^ (0 - ((k3456[threadIdx.x] >> 7) & 1)) ^ tw1[31];
  aa[threadIdx.x] = sbox(aa[threadIdx.x]);

  cc.b0 ^= bb.b0 ^ 0xaaaaaaaa                     ^ tw1[16];
  cc.b1 ^= bb.b1 ^ 0xcccccccc                     ^ tw1[17];
  cc.b2 ^= bb.b2 ^ 0xf0f0f0f0                     ^ tw1[18];
  cc.b3 ^= bb.b3 ^ 0xff00ff00                     ^ tw1[19];
  cc.b4 ^= bb.b4 ^ 0xffff0000                     ^ tw1[20];
  cc.b5 ^= bb.b5 ^ (0 -  (threadIdx.x       & 1)) ^ tw1[21];
  cc.b6 ^= bb.b6 ^ (0 - ((threadIdx.x >> 1) & 1)) ^ tw1[22];
  cc.b7 ^= bb.b7 ^ (0 - ((threadIdx.x >> 2) & 1)) ^ tw1[23];
  cc = sbox(cc);

  bb.b0 ^= aa[threadIdx.x].b0 ^ cc.b0 ^ key_12[8]  ^ tw1[8];
  bb.b1 ^= aa[threadIdx.x].b1 ^ cc.b1 ^ key_12[9]  ^ tw1[9];
  bb.b2 ^= aa[threadIdx.x].b2 ^ cc.b2 ^ key_12[10] ^ tw1[10];
  bb.b3 ^= aa[threadIdx.x].b3 ^ cc.b3 ^ key_12[11] ^ tw1[11];
  bb.b4 ^= aa[threadIdx.x].b4 ^ cc.b4 ^ key_12[12] ^ tw1[12];
  bb.b5 ^= aa[threadIdx.x].b5 ^ cc.b5 ^ key_12[13] ^ tw1[13];
  bb.b6 ^= aa[threadIdx.x].b6 ^ cc.b6 ^ key_12[14] ^ tw1[14];
  bb.b7 ^= aa[threadIdx.x].b7 ^ cc.b7 ^ key_12[15] ^ tw1[15];
  bb = sbox(bb);

  /* Round 6. */
  aa[threadIdx.x].b0 ^= bb.b0 ^ key_12[0] ^ tw1[0];
  aa[threadIdx.x].b1 ^= bb.b1 ^ key_12[1] ^ tw1[1];
  aa[threadIdx.x].b2 ^= bb.b2 ^ key_12[2] ^ tw1[2];
  aa[threadIdx.x].b3 ^= bb.b3 ^ key_12[3] ^ tw1[3];
  aa[threadIdx.x].b4 ^= bb.b4 ^ key_12[4] ^ tw1[4];
  aa[threadIdx.x].b5 ^= bb.b5 ^ key_12[5] ^ tw1[5];
  aa[threadIdx.x].b6 ^= bb.b6 ^ key_12[6] ^ tw1[6];
  aa[threadIdx.x].b7 ^= bb.b7 ^ key_12[7] ^ tw1[7];
  aa[threadIdx.x] = sbox(aa[threadIdx.x]);

  cc.b0 ^= bb.b0 ^ (0 - ((k3456[threadIdx.x] >> 24) & 1)) ^ tw1[56];
  cc.b1 ^= bb.b1 ^ (0 - ((k3456[threadIdx.x] >> 25) & 1)) ^ tw1[57];
  cc.b2 ^= bb.b2 ^ (0 - ((k3456[threadIdx.x] >> 26) & 1)) ^ tw1[58];
  cc.b3 ^= bb.b3 ^ (0 - ((k3456[threadIdx.x] >> 27) & 1)) ^ tw1[59];
  cc.b4 ^= bb.b4 ^ (0 - ((k3456[threadIdx.x] >> 28) & 1)) ^ tw1[60];
  cc.b5 ^= bb.b5 ^ (0 - ((k3456[threadIdx.x] >> 29) & 1)) ^ tw1[61];
  cc.b6 ^= bb.b6 ^ (0 - ((k3456[threadIdx.x] >> 30) & 1)) ^ tw1[62];
  cc.b7 ^= bb.b7 ^ (0 - ((k3456[threadIdx.x] >> 31) & 1)) ^ tw1[63];
  cc = sbox(cc);

  bb.b0 ^= aa[threadIdx.x].b0 ^ cc.b0 ^ (0 - ((k3456[threadIdx.x] >> 16) & 1)) ^ tw1[48];
  bb.b1 ^= aa[threadIdx.x].b1 ^ cc.b1 ^ (0 - ((k3456[threadIdx.x] >> 17) & 1)) ^ tw1[49];
  bb.b2 ^= aa[threadIdx.x].b2 ^ cc.b2 ^ (0 - ((k3456[threadIdx.x] >> 18) & 1)) ^ tw1[50];
  bb.b3 ^= aa[threadIdx.x].b3 ^ cc.b3 ^ (0 - ((k3456[threadIdx.x] >> 19) & 1)) ^ tw1[51];
  bb.b4 ^= aa[threadIdx.x].b4 ^ cc.b4 ^ (0 - ((k3456[threadIdx.x] >> 20) & 1)) ^ tw1[52];
  bb.b5 ^= aa[threadIdx.x].b5 ^ cc.b5 ^ (0 - ((k3456[threadIdx.x] >> 21) & 1)) ^ tw1[53];
  bb.b6 ^= aa[threadIdx.x].b6 ^ cc.b6 ^ (0 - ((k3456[threadIdx.x] >> 22) & 1)) ^ tw1[54];
  bb.b7 ^= aa[threadIdx.x].b7 ^ cc.b7 ^ (0 - ((k3456[threadIdx.x] >> 23) & 1)) ^ tw1[55];
  bb = sbox(bb);

  /* Round 7. */
  if (rounds > 6) {
    aa[threadIdx.x].b0 ^= bb.b0 ^ (0 - ((k3456[threadIdx.x] >> 8) & 1))  ^ tw1[40];
    aa[threadIdx.x].b1 ^= bb.b1 ^ (0 - ((k3456[threadIdx.x] >> 9) & 1))  ^ tw1[41];
    aa[threadIdx.x].b2 ^= bb.b2 ^ (0 - ((k3456[threadIdx.x] >> 10) & 1)) ^ tw1[42];
    aa[threadIdx.x].b3 ^= bb.b3 ^ (0 - ((k3456[threadIdx.x] >> 11) & 1)) ^ tw1[43];
    aa[threadIdx.x].b4 ^= bb.b4 ^ (0 - ((k3456[threadIdx.x] >> 12) & 1)) ^ tw1[44];
    aa[threadIdx.x].b5 ^= bb.b5 ^ (0 - ((k3456[threadIdx.x] >> 13) & 1)) ^ tw1[45];
    aa[threadIdx.x].b6 ^= bb.b6 ^ (0 - ((k3456[threadIdx.x] >> 14) & 1)) ^ tw1[46];
    aa[threadIdx.x].b7 ^= bb.b7 ^ (0 - ((k3456[threadIdx.x] >> 15) & 1)) ^ tw1[47];
    aa[threadIdx.x] = sbox(aa[threadIdx.x]);

    cc.b0 ^= bb.b0 ^ (0 - ((k3456[threadIdx.x] >> 0) & 1)) ^ tw1[32];
    cc.b1 ^= bb.b1 ^ (0 - ((k3456[threadIdx.x] >> 1) & 1)) ^ tw1[33];
    cc.b2 ^= bb.b2 ^ (0 - ((k3456[threadIdx.x] >> 2) & 1)) ^ tw1[34];
    cc.b3 ^= bb.b3 ^ (0 - ((k3456[threadIdx.x] >> 3) & 1)) ^ tw1[35];
    cc.b4 ^= bb.b4 ^ (0 - ((k3456[threadIdx.x] >> 4) & 1)) ^ tw1[36];
    cc.b5 ^= bb.b5 ^ (0 - ((k3456[threadIdx.x] >> 5) & 1)) ^ tw1[37];
    cc.b6 ^= bb.b6 ^ (0 - ((k3456[threadIdx.x] >> 6) & 1)) ^ tw1[38];
    cc.b7 ^= bb.b7 ^ (0 - ((k3456[threadIdx.x] >> 7) & 1)) ^ tw1[39];
    cc = sbox(cc);

    bb.b0 ^= aa[threadIdx.x].b0 ^ cc.b0 ^ 0xaaaaaaaa                     ^ tw1[24];
    bb.b1 ^= aa[threadIdx.x].b1 ^ cc.b1 ^ 0xcccccccc                     ^ tw1[25];
    bb.b2 ^= aa[threadIdx.x].b2 ^ cc.b2 ^ 0xf0f0f0f0                     ^ tw1[26];
    bb.b3 ^= aa[threadIdx.x].b3 ^ cc.b3 ^ 0xff00ff00                     ^ tw1[27];
    bb.b4 ^= aa[threadIdx.x].b4 ^ cc.b4 ^ 0xffff0000                     ^ tw1[28];
    bb.b5 ^= aa[threadIdx.x].b5 ^ cc.b5 ^ (0 -  (threadIdx.x       & 1)) ^ tw1[29];
    bb.b6 ^= aa[threadIdx.x].b6 ^ cc.b6 ^ (0 - ((threadIdx.x >> 1) & 1)) ^ tw1[30];
    bb.b7 ^= aa[threadIdx.x].b7 ^ cc.b7 ^ (0 - ((threadIdx.x >> 2) & 1)) ^ tw1[31];
    bb = sbox(bb);
  }

  /* Round 8. */
  if (rounds > 7) {
    aa[threadIdx.x].b0 ^= bb.b0 ^ key_12[8]  ^ tw1[16];
    aa[threadIdx.x].b1 ^= bb.b1 ^ key_12[9]  ^ tw1[17];
    aa[threadIdx.x].b2 ^= bb.b2 ^ key_12[10] ^ tw1[18];
    aa[threadIdx.x].b3 ^= bb.b3 ^ key_12[11] ^ tw1[19];
    aa[threadIdx.x].b4 ^= bb.b4 ^ key_12[12] ^ tw1[20];
    aa[threadIdx.x].b5 ^= bb.b5 ^ key_12[13] ^ tw1[21];
    aa[threadIdx.x].b6 ^= bb.b6 ^ key_12[14] ^ tw1[22];
    aa[threadIdx.x].b7 ^= bb.b7 ^ key_12[15] ^ tw1[23];
    aa[threadIdx.x] = sbox(aa[threadIdx.x]);

    cc.b0 ^= bb.b0 ^ key_12[0] ^ tw1[8];
    cc.b1 ^= bb.b1 ^ key_12[1] ^ tw1[9];
    cc.b2 ^= bb.b2 ^ key_12[2] ^ tw1[10];
    cc.b3 ^= bb.b3 ^ key_12[3] ^ tw1[11];
    cc.b4 ^= bb.b4 ^ key_12[4] ^ tw1[12];
    cc.b5 ^= bb.b5 ^ key_12[5] ^ tw1[13];
    cc.b6 ^= bb.b6 ^ key_12[6] ^ tw1[14];
    cc.b7 ^= bb.b7 ^ key_12[7] ^ tw1[15];
    cc = sbox(cc);

    bb.b0 ^= aa[threadIdx.x].b0 ^ cc.b0 ^ (0 - ((k3456[threadIdx.x] >> 24) & 1)) ^ tw1[0];
    bb.b1 ^= aa[threadIdx.x].b1 ^ cc.b1 ^ (0 - ((k3456[threadIdx.x] >> 25) & 1)) ^ tw1[1];
    bb.b2 ^= aa[threadIdx.x].b2 ^ cc.b2 ^ (0 - ((k3456[threadIdx.x] >> 26) & 1)) ^ tw1[2];
    bb.b3 ^= aa[threadIdx.x].b3 ^ cc.b3 ^ (0 - ((k3456[threadIdx.x] >> 27) & 1)) ^ tw1[3];
    bb.b4 ^= aa[threadIdx.x].b4 ^ cc.b4 ^ (0 - ((k3456[threadIdx.x] >> 28) & 1)) ^ tw1[4];
    bb.b5 ^= aa[threadIdx.x].b5 ^ cc.b5 ^ (0 - ((k3456[threadIdx.x] >> 29) & 1)) ^ tw1[5];
    bb.b6 ^= aa[threadIdx.x].b6 ^ cc.b6 ^ (0 - ((k3456[threadIdx.x] >> 30) & 1)) ^ tw1[6];
    bb.b7 ^= aa[threadIdx.x].b7 ^ cc.b7 ^ (0 - ((k3456[threadIdx.x] >> 31) & 1)) ^ tw1[7];
    bb = sbox(bb);
  }

  int cmp;
  cmp  = cc.b0 ^ ct1[0];
  cmp |= cc.b1 ^ ct1[1];
  cmp |= cc.b2 ^ ct1[2];
  cmp |= cc.b3 ^ ct1[3];
  cmp |= cc.b4 ^ ct1[4];
  cmp |= cc.b5 ^ ct1[5];
  cmp |= cc.b6 ^ ct1[6];
  cmp |= cc.b7 ^ ct1[7];
  cmp |= bb.b0 ^ ct1[8];
  cmp |= bb.b1 ^ ct1[9];
  cmp |= bb.b2 ^ ct1[10];
  cmp |= bb.b3 ^ ct1[11];
  cmp |= bb.b4 ^ ct1[12];
  cmp |= bb.b5 ^ ct1[13];
  cmp |= bb.b6 ^ ct1[14];
  cmp |= bb.b7 ^ ct1[15];
  cmp |= aa[threadIdx.x].b0 ^ ct1[16];
  cmp |= aa[threadIdx.x].b1 ^ ct1[17];
  cmp |= aa[threadIdx.x].b2 ^ ct1[18];
  cmp |= aa[threadIdx.x].b3 ^ ct1[19];
  cmp |= aa[threadIdx.x].b4 ^ ct1[20];
  cmp |= aa[threadIdx.x].b5 ^ ct1[21];
  cmp |= aa[threadIdx.x].b6 ^ ct1[22];
  cmp |= aa[threadIdx.x].b7 ^ ct1[23];
  cmp = ~cmp;

  found[threadIdx.x] = 0;

  __syncthreads();

  /* Put matches in shared memory. */
  if (cmp != 0) {
    int resultp = atomicAdd_block(found, __popc(cmp)) * 2 + 2;
    while (cmp != 0) {
      int low5 = __ffs(cmp) - 1;
      cmp ^= 1 << low5;
      found[resultp] = k3456[threadIdx.x];
      found[resultp + 1] = ((threadIdx.x & 7) << 5) | low5;
      resultp += 2;
    }
  }

  __syncthreads();

  if (found[0] == 0) {
    return;
  }

  /* Get global memory offset for matches found in block. */
  if (threadIdx.x == 0) {
    found[1] = atomicAdd(out, found[0]) * 2 + 1;
  }

  __syncthreads();

  /* Copy matches to global memory. */
  if (threadIdx.x < (found[1] * 2)) {
    int ptr = found[1] + threadIdx.x;
    out[ptr] = found[threadIdx.x + 2];
  }
}

__launch_bounds__(1024, 1)
__global__ void find_candidates(int *ret, int offset) {

  volatile __shared__ int pt1[24];
  volatile __shared__ int pt2[24];
  volatile __shared__ int tw1[64];
  volatile __shared__ int tw2[64];
  volatile __shared__ int key_12[16];
  __shared__ int found[1024];

  if (threadIdx.x < 16) {
    key_12[threadIdx.x] = key_c[threadIdx.x + offset * 16];
  }
  if (threadIdx.x < 24) {
    pt1[threadIdx.x] = pt1_c[threadIdx.x + offset * 24];
    pt2[threadIdx.x] = pt2_c[threadIdx.x + offset * 24];
  }
  if (threadIdx.x < 64) {
    tw1[threadIdx.x] = tw1_c[threadIdx.x + offset * 64];
    tw2[threadIdx.x] = tw2_c[threadIdx.x + offset * 64];
  }

  __syncthreads();

  int key3 = key3_c[blockIdx.x >> 9];

  eightbits aa, bb, cc;

  /* PT1: Round 1. */
  bb.b0 = pt1[8];
  bb.b1 = pt1[9];
  bb.b2 = pt1[10];
  bb.b3 = pt1[11];
  bb.b4 = pt1[12];
  bb.b5 = pt1[13];
  bb.b6 = pt1[14];
  bb.b7 = pt1[15];
  aa.b0 = pt1[16] ^ bb.b0 ^ key_12[8]  ^ tw1[56];
  aa.b1 = pt1[17] ^ bb.b1 ^ key_12[9]  ^ tw1[57];
  aa.b2 = pt1[18] ^ bb.b2 ^ key_12[10] ^ tw1[58];
  aa.b3 = pt1[19] ^ bb.b3 ^ key_12[11] ^ tw1[59];
  aa.b4 = pt1[20] ^ bb.b4 ^ key_12[12] ^ tw1[60];
  aa.b5 = pt1[21] ^ bb.b5 ^ key_12[13] ^ tw1[61];
  aa.b6 = pt1[22] ^ bb.b6 ^ key_12[14] ^ tw1[62];
  aa.b7 = pt1[23] ^ bb.b7 ^ key_12[15] ^ tw1[63];
  aa = sbox(aa);

  cc.b0 = pt1[0]  ^ bb.b0 ^ key_12[0] ^ tw1[48];
  cc.b1 = pt1[1]  ^ bb.b1 ^ key_12[1] ^ tw1[49];
  cc.b2 = pt1[2]  ^ bb.b2 ^ key_12[2] ^ tw1[50];
  cc.b3 = pt1[3]  ^ bb.b3 ^ key_12[3] ^ tw1[51];
  cc.b4 = pt1[4]  ^ bb.b4 ^ key_12[4] ^ tw1[52];
  cc.b5 = pt1[5]  ^ bb.b5 ^ key_12[5] ^ tw1[53];
  cc.b6 = pt1[6]  ^ bb.b6 ^ key_12[6] ^ tw1[54];
  cc.b7 = pt1[7]  ^ bb.b7 ^ key_12[7] ^ tw1[55];
  cc = sbox(cc);

  bb.b0 ^= aa.b0  ^ cc.b0 ^ (0 - ((key3 >> 0) & 1)) ^ tw1[40];
  bb.b1 ^= aa.b1  ^ cc.b1 ^ (0 - ((key3 >> 1) & 1)) ^ tw1[41];
  bb.b2 ^= aa.b2  ^ cc.b2 ^ (0 - ((key3 >> 2) & 1)) ^ tw1[42];
  bb.b3 ^= aa.b3  ^ cc.b3 ^ (0 - ((key3 >> 3) & 1)) ^ tw1[43];
  bb.b4 ^= aa.b4  ^ cc.b4 ^ (0 - ((key3 >> 4) & 1)) ^ tw1[44];
  bb.b5 ^= aa.b5  ^ cc.b5 ^ (0 - ((key3 >> 5) & 1)) ^ tw1[45];
  bb.b6 ^= aa.b6  ^ cc.b6 ^ (0 - ((key3 >> 6) & 1)) ^ tw1[46];
  bb.b7 ^= aa.b7  ^ cc.b7 ^ (0 - ((key3 >> 7) & 1)) ^ tw1[47];
  bb = sbox(bb);

  /* PT1: Round 2. */
  aa.b0 ^= bb.b0 ^ (0 - ((blockIdx.x  >> 1) & 1)) ^ tw1[32];
  aa.b1 ^= bb.b1 ^ (0 - ((blockIdx.x  >> 2) & 1)) ^ tw1[33];
  aa.b2 ^= bb.b2 ^ (0 - ((blockIdx.x  >> 3) & 1)) ^ tw1[34];
  aa.b3 ^= bb.b3 ^ (0 - ((blockIdx.x  >> 4) & 1)) ^ tw1[35];
  aa.b4 ^= bb.b4 ^ (0 - ((blockIdx.x  >> 5) & 1)) ^ tw1[36];
  aa.b5 ^= bb.b5 ^ (0 - ((blockIdx.x  >> 6) & 1)) ^ tw1[37];
  aa.b6 ^= bb.b6 ^ (0 - ((blockIdx.x  >> 7) & 1)) ^ tw1[38];
  aa.b7 ^= bb.b7 ^ (0 - ((blockIdx.x  >> 8) & 1)) ^ tw1[39];
  aa = sbox(aa);

  cc.b0 ^= bb.b0 ^ (0 - ((threadIdx.x >> 3) & 1)) ^ tw1[24];
  cc.b1 ^= bb.b1 ^ (0 - ((threadIdx.x >> 4) & 1)) ^ tw1[25];
  cc.b2 ^= bb.b2 ^ (0 - ((threadIdx.x >> 5) & 1)) ^ tw1[26];
  cc.b3 ^= bb.b3 ^ (0 - ((threadIdx.x >> 6) & 1)) ^ tw1[27];
  cc.b4 ^= bb.b4 ^ (0 - ((threadIdx.x >> 7) & 1)) ^ tw1[28];
  cc.b5 ^= bb.b5 ^ (0 - ((threadIdx.x >> 8) & 1)) ^ tw1[29];
  cc.b6 ^= bb.b6 ^ (0 - ((threadIdx.x >> 9) & 1)) ^ tw1[30];
  cc.b7 ^= bb.b7 ^ (0 -  (blockIdx.x        & 1)) ^ tw1[31];
  cc = sbox(cc);

  bb.b0 ^= aa.b0 ^ cc.b0 ^ 0xaaaaaaaa                     ^ tw1[16];
  bb.b1 ^= aa.b1 ^ cc.b1 ^ 0xcccccccc                     ^ tw1[17];
  bb.b2 ^= aa.b2 ^ cc.b2 ^ 0xf0f0f0f0                     ^ tw1[18];
  bb.b3 ^= aa.b3 ^ cc.b3 ^ 0xff00ff00                     ^ tw1[19];
  bb.b4 ^= aa.b4 ^ cc.b4 ^ 0xffff0000                     ^ tw1[20];
  bb.b5 ^= aa.b5 ^ cc.b5 ^ (0 - ((threadIdx.x >> 0) & 1)) ^ tw1[21];
  bb.b6 ^= aa.b6 ^ cc.b6 ^ (0 - ((threadIdx.x >> 1) & 1)) ^ tw1[22];
  bb.b7 ^= aa.b7 ^ cc.b7 ^ (0 - ((threadIdx.x >> 2) & 1)) ^ tw1[23];
  bb = sbox(bb);

  /* PT1: Round 3c. */
  cc.b0 ^= bb.b0 ^ key_12[8]  ^ tw1[0];
  cc.b1 ^= bb.b1 ^ key_12[9]  ^ tw1[1];
  cc.b2 ^= bb.b2 ^ key_12[10] ^ tw1[2];
  cc.b3 ^= bb.b3 ^ key_12[11] ^ tw1[3];
  cc.b4 ^= bb.b4 ^ key_12[12] ^ tw1[4];
  cc.b5 ^= bb.b5 ^ key_12[13] ^ tw1[5];
  cc.b6 ^= bb.b6 ^ key_12[14] ^ tw1[6];
  cc.b7 ^= bb.b7 ^ key_12[15] ^ tw1[7];
  cc = sbox(cc);

  eightbits xx;
  xx.b0 = cc.b0 ^ tw1[24] ^ tw2[24];
  xx.b1 = cc.b1 ^ tw1[25] ^ tw2[25];
  xx.b2 = cc.b2 ^ tw1[26] ^ tw2[26];
  xx.b3 = cc.b3 ^ tw1[27] ^ tw2[27];
  xx.b4 = cc.b4 ^ tw1[28] ^ tw2[28];
  xx.b5 = cc.b5 ^ tw1[29] ^ tw2[29];
  xx.b6 = cc.b6 ^ tw1[30] ^ tw2[30];
  xx.b7 = cc.b7 ^ tw1[31] ^ tw2[31];

  /* PT2: Round 1. */
  bb.b0 = pt2[8];
  bb.b1 = pt2[9];
  bb.b2 = pt2[10];
  bb.b3 = pt2[11];
  bb.b4 = pt2[12];
  bb.b5 = pt2[13];
  bb.b6 = pt2[14];
  bb.b7 = pt2[15];
  aa.b0 = pt2[16] ^ bb.b0 ^ key_12[8]  ^ tw2[56];
  aa.b1 = pt2[17] ^ bb.b1 ^ key_12[9]  ^ tw2[57];
  aa.b2 = pt2[18] ^ bb.b2 ^ key_12[10] ^ tw2[58];
  aa.b3 = pt2[19] ^ bb.b3 ^ key_12[11] ^ tw2[59];
  aa.b4 = pt2[20] ^ bb.b4 ^ key_12[12] ^ tw2[60];
  aa.b5 = pt2[21] ^ bb.b5 ^ key_12[13] ^ tw2[61];
  aa.b6 = pt2[22] ^ bb.b6 ^ key_12[14] ^ tw2[62];
  aa.b7 = pt2[23] ^ bb.b7 ^ key_12[15] ^ tw2[63];
  aa = sbox(aa);

  cc.b0 = pt2[0]  ^ bb.b0 ^ key_12[0] ^ tw2[48];
  cc.b1 = pt2[1]  ^ bb.b1 ^ key_12[1] ^ tw2[49];
  cc.b2 = pt2[2]  ^ bb.b2 ^ key_12[2] ^ tw2[50];
  cc.b3 = pt2[3]  ^ bb.b3 ^ key_12[3] ^ tw2[51];
  cc.b4 = pt2[4]  ^ bb.b4 ^ key_12[4] ^ tw2[52];
  cc.b5 = pt2[5]  ^ bb.b5 ^ key_12[5] ^ tw2[53];
  cc.b6 = pt2[6]  ^ bb.b6 ^ key_12[6] ^ tw2[54];
  cc.b7 = pt2[7]  ^ bb.b7 ^ key_12[7] ^ tw2[55];
  cc = sbox(cc);

  bb.b0 ^= aa.b0  ^ cc.b0 ^ (0 - ((key3 >> 0) & 1)) ^ tw2[40];
  bb.b1 ^= aa.b1  ^ cc.b1 ^ (0 - ((key3 >> 1) & 1)) ^ tw2[41];
  bb.b2 ^= aa.b2  ^ cc.b2 ^ (0 - ((key3 >> 2) & 1)) ^ tw2[42];
  bb.b3 ^= aa.b3  ^ cc.b3 ^ (0 - ((key3 >> 3) & 1)) ^ tw2[43];
  bb.b4 ^= aa.b4  ^ cc.b4 ^ (0 - ((key3 >> 4) & 1)) ^ tw2[44];
  bb.b5 ^= aa.b5  ^ cc.b5 ^ (0 - ((key3 >> 5) & 1)) ^ tw2[45];
  bb.b6 ^= aa.b6  ^ cc.b6 ^ (0 - ((key3 >> 6) & 1)) ^ tw2[46];
  bb.b7 ^= aa.b7  ^ cc.b7 ^ (0 - ((key3 >> 7) & 1)) ^ tw2[47];
  bb = sbox(bb);

  /* PT2: Round 2. */
  aa.b0 ^= bb.b0 ^ (0 - ((blockIdx.x  >> 1) & 1)) ^ tw2[32];
  aa.b1 ^= bb.b1 ^ (0 - ((blockIdx.x  >> 2) & 1)) ^ tw2[33];
  aa.b2 ^= bb.b2 ^ (0 - ((blockIdx.x  >> 3) & 1)) ^ tw2[34];
  aa.b3 ^= bb.b3 ^ (0 - ((blockIdx.x  >> 4) & 1)) ^ tw2[35];
  aa.b4 ^= bb.b4 ^ (0 - ((blockIdx.x  >> 5) & 1)) ^ tw2[36];
  aa.b5 ^= bb.b5 ^ (0 - ((blockIdx.x  >> 6) & 1)) ^ tw2[37];
  aa.b6 ^= bb.b6 ^ (0 - ((blockIdx.x  >> 7) & 1)) ^ tw2[38];
  aa.b7 ^= bb.b7 ^ (0 - ((blockIdx.x  >> 8) & 1)) ^ tw2[39];
  aa = sbox(aa);

  cc.b0 ^= bb.b0 ^ (0 - ((threadIdx.x >> 3) & 1)) ^ tw2[24];
  cc.b1 ^= bb.b1 ^ (0 - ((threadIdx.x >> 4) & 1)) ^ tw2[25];
  cc.b2 ^= bb.b2 ^ (0 - ((threadIdx.x >> 5) & 1)) ^ tw2[26];
  cc.b3 ^= bb.b3 ^ (0 - ((threadIdx.x >> 6) & 1)) ^ tw2[27];
  cc.b4 ^= bb.b4 ^ (0 - ((threadIdx.x >> 7) & 1)) ^ tw2[28];
  cc.b5 ^= bb.b5 ^ (0 - ((threadIdx.x >> 8) & 1)) ^ tw2[29];
  cc.b6 ^= bb.b6 ^ (0 - ((threadIdx.x >> 9) & 1)) ^ tw2[30];
  cc.b7 ^= bb.b7 ^ (0 -  (blockIdx.x        & 1)) ^ tw2[31];
  cc = sbox(cc);

  bb.b0 ^= aa.b0 ^ cc.b0 ^ 0xaaaaaaaa                     ^ tw2[16];
  bb.b1 ^= aa.b1 ^ cc.b1 ^ 0xcccccccc                     ^ tw2[17];
  bb.b2 ^= aa.b2 ^ cc.b2 ^ 0xf0f0f0f0                     ^ tw2[18];
  bb.b3 ^= aa.b3 ^ cc.b3 ^ 0xff00ff00                     ^ tw2[19];
  bb.b4 ^= aa.b4 ^ cc.b4 ^ 0xffff0000                     ^ tw2[20];
  bb.b5 ^= aa.b5 ^ cc.b5 ^ (0 - ((threadIdx.x >> 0) & 1)) ^ tw2[21];
  bb.b6 ^= aa.b6 ^ cc.b6 ^ (0 - ((threadIdx.x >> 1) & 1)) ^ tw2[22];
  bb.b7 ^= aa.b7 ^ cc.b7 ^ (0 - ((threadIdx.x >> 2) & 1)) ^ tw2[23];
  bb = sbox(bb);

  /* PT2: Round 3c. */
  cc.b0 ^= bb.b0 ^ key_12[8]  ^ tw2[0];
  cc.b1 ^= bb.b1 ^ key_12[9]  ^ tw2[1];
  cc.b2 ^= bb.b2 ^ key_12[10] ^ tw2[2];
  cc.b3 ^= bb.b3 ^ key_12[11] ^ tw2[3];
  cc.b4 ^= bb.b4 ^ key_12[12] ^ tw2[4];
  cc.b5 ^= bb.b5 ^ key_12[13] ^ tw2[5];
  cc.b6 ^= bb.b6 ^ key_12[14] ^ tw2[6];
  cc.b7 ^= bb.b7 ^ key_12[15] ^ tw2[7];
  cc = sbox(cc);

  int rr;
  rr  = cc.b0 ^ xx.b0;
  rr |= cc.b1 ^ xx.b1;
  rr |= cc.b2 ^ xx.b2;
  rr |= cc.b3 ^ xx.b3;
  rr |= cc.b4 ^ xx.b4;
  rr |= cc.b5 ^ xx.b5;
  rr |= cc.b6 ^ xx.b6;
  rr |= cc.b7 ^ xx.b7;
  rr = ~rr;

  found[threadIdx.x] = 0;

  __syncthreads();

  if (rr != 0) {
    int ptr = atomicAdd_block(found, __popc(rr));
    int k3456 = (key3 << 24) | ((blockIdx.x & 0x1ff) << 15) | (threadIdx.x << 5);
    while (rr != 0) {
      int low5 = __ffs(rr) - 1;
      rr ^= 1 << low5;
      found[ptr + 2] = k3456 | low5;
      ptr += 1;
    }
  }

  __syncthreads();

  if (found[0] == 0) {
    return;
  }

  if (threadIdx.x == 0) {
    found[1] = atomicAdd(ret, found[0]) + 1;
  }

  __syncthreads();

  if (threadIdx.x < found[0]) {
    int ptr = found[1] + threadIdx.x;
    ret[ptr] = found[threadIdx.x + 2];
  }
}

/* Host functions. */

void list_cuda_devices() {
  int count = -1;
  hipError_t err;
  err = hipGetDeviceCount(&count);
  if (err == 30 || count == 0) {
    printf("No CUDA devices found.\n");
    return;
  }
  if (err != 0) {
    fprintf(stderr, "Error: hipGetDeviceCount returned error %d.\n", err);
    return;
  }
  for (int i = 0; i < count; i++) {
    hipDeviceProp_t prop;
    err = hipGetDeviceProperties(&prop, i);
    if (err != 0) {
      printf("Error when getting properties for device %d.\n", i);
    } else {
      printf("CUDA Device %d: %s\n", i, prop.name);
    }
  }
}

int get_num_cuda_devices() {
  int count;
  hipError_t err;
  err = hipGetDeviceCount(&count);
  if (err == 30) {
    return 0;
  }
  if (err != 0) {
    return -1;
  }
  return count;
}

#define CUDA_FAST_RETURN_ON_ERROR(err, stream, ret, dev_mem)\
  if (err != hipSuccess) {\
    fprintf(stderr, "CUDA error. (%s:%d)\n", __FILE__, __LINE__);\
    if (stream != NULL) { hipStreamDestroy(stream); }\
    if (ret != NULL) { hipHostFree(ret); }\
    if (dev_mem != NULL) { hipFree(dev_mem); }\
    return;\
  }

void cuda_fast(worker_param_t params, uint32_t threadid, uint32_t cuda_device) {

  uint32_t *ret = NULL;
  int32_t *dev_mem = NULL;
  int32_t *dev_mem2 = NULL;
  hipStream_t stream = NULL;

  hipError_t err = hipSetDevice(cuda_device);
  CUDA_FAST_RETURN_ON_ERROR(err, stream, ret, dev_mem);

  /* Allocate host memory. */
  err = hipHostAlloc(&ret, sizeof(int32_t) * 0x2000000, hipHostMallocDefault);
  CUDA_FAST_RETURN_ON_ERROR(err, stream, ret, dev_mem);
  assert(ret != NULL);

  /* Allocate device memory. */

  err = hipMalloc(&dev_mem, sizeof(int32_t) * 0x2000000);
  CUDA_FAST_RETURN_ON_ERROR(err, stream, ret, dev_mem);
  err = hipMalloc(&dev_mem2, sizeof(int32_t) * 1000);
  CUDA_FAST_RETURN_ON_ERROR(err, stream, ret, dev_mem);
  assert(dev_mem != NULL);
  assert(dev_mem2 != NULL);

  err = hipStreamCreate(&stream);
  CUDA_FAST_RETURN_ON_ERROR(err, stream, ret, dev_mem);

  uint32_t k12;
  pair_t *pair;
  int32_t hi_key_bits[24];
  int32_t pt1_bits[24];
  int32_t pt2_bits[24];
  int32_t ct1_bits[24];
  int32_t tw1_bits[64];
  int32_t tw2_bits[64];

  while (!g_exit && get_next_678(threadid, &k12, &pair)) {

    /* Set plaintext, tweak and high key bits. */
    for (int bit = 0; bit < 24; bit++) {
      hi_key_bits[bit] = 0 - ((k12 >> bit) & 1);
      pt1_bits[bit]    = 0 - ((pair->t1.pt >> bit) & 1);
      pt2_bits[bit]    = 0 - ((pair->t2.pt >> bit) & 1);
      ct1_bits[bit]    = 0 - ((pair->t1.ct >> bit) & 1);
    }
    for (int bit = 0; bit < 64; bit++) {
      tw1_bits[bit] = 0 - ((pair->t1.tw >> bit) & 1);
      tw2_bits[bit] = 0 - ((pair->t2.tw >> bit) & 1);
    }

    err = hipMemcpyToSymbolAsync(HIP_SYMBOL(key_c), hi_key_bits, sizeof(int32_t) * 24, 0,
        hipMemcpyHostToDevice, stream);
    CUDA_FAST_RETURN_ON_ERROR(err, stream, ret, dev_mem);
    err = hipMemcpyToSymbolAsync(HIP_SYMBOL(pt1_c), pt1_bits,    sizeof(int32_t) * 24, 0,
        hipMemcpyHostToDevice, stream);
    CUDA_FAST_RETURN_ON_ERROR(err, stream, ret, dev_mem);
    err = hipMemcpyToSymbolAsync(HIP_SYMBOL(pt2_c), pt2_bits,    sizeof(int32_t) * 24, 0,
        hipMemcpyHostToDevice, stream);
    CUDA_FAST_RETURN_ON_ERROR(err, stream, ret, dev_mem);
    err = hipMemcpyToSymbolAsync(HIP_SYMBOL(ct1_c), ct1_bits,    sizeof(int32_t) * 24, 0,
        hipMemcpyHostToDevice, stream);
    CUDA_FAST_RETURN_ON_ERROR(err, stream, ret, dev_mem);
    err = hipMemcpyToSymbolAsync(HIP_SYMBOL(tw1_c), tw1_bits,    sizeof(int32_t) * 64, 0,
        hipMemcpyHostToDevice, stream);
    CUDA_FAST_RETURN_ON_ERROR(err, stream, ret, dev_mem);
    err = hipMemcpyToSymbolAsync(HIP_SYMBOL(tw2_c), tw2_bits,    sizeof(int32_t) * 64, 0,
        hipMemcpyHostToDevice, stream);
    CUDA_FAST_RETURN_ON_ERROR(err, stream, ret, dev_mem);
    err = hipMemcpyToSymbolAsync(HIP_SYMBOL(key3_c), pair->k3,   sizeof(int32_t) * 256, 0,
        hipMemcpyHostToDevice, stream);

    err = hipMemsetAsync(dev_mem, 0, sizeof(int32_t) * 1, stream);
    CUDA_FAST_RETURN_ON_ERROR(err, stream, ret, dev_mem);
    err = hipMemsetAsync(dev_mem2, 0, sizeof(int32_t) * 1, stream);
    CUDA_FAST_RETURN_ON_ERROR(err, stream, ret, dev_mem);

    find_candidates<<<512 * pair->num_k3, 1024, 0, stream>>>(dev_mem, 0);
    err = hipMemcpyAsync(ret, dev_mem, sizeof(int32_t) * 1, hipMemcpyDefault, stream);
    CUDA_FAST_RETURN_ON_ERROR(err, stream, ret, dev_mem);

    err = hipStreamSynchronize(stream);
    CUDA_FAST_RETURN_ON_ERROR(err, stream, ret, dev_mem);

    int num_blocks = (ret[0] + 129) / 128;
    switch (params.nrounds) {
      case 6:
        test_candidates<6><<<num_blocks, 1024, 0, stream>>>(dev_mem, dev_mem2, ret[0], 0);
        break;
      case 7:
        test_candidates<7><<<num_blocks, 1024, 0, stream>>>(dev_mem, dev_mem2, ret[0], 0);
        break;
      case 8:
        test_candidates<8><<<num_blocks, 1024, 0, stream>>>(dev_mem, dev_mem2, ret[0], 0);
        break;
      default:
        assert(0);
    }
    err = hipMemcpyAsync(ret, dev_mem2, sizeof(int32_t) * 1000, hipMemcpyDefault, stream);
    CUDA_FAST_RETURN_ON_ERROR(err, stream, ret, dev_mem);

    err = hipStreamSynchronize(stream);
    CUDA_FAST_RETURN_ON_ERROR(err, stream, ret, dev_mem);

    for (int i = 0; i < ret[0]; i++) {
      uint64_t key = ((uint64_t)k12 << 40) | ((uint64_t)ret[i * 2 + 1] << 8) | ret[i * 2 + 2];
      if (test_key(params.nrounds, key, params.tuples, params.num_tuples)) {
        found_key(key);
      }
    }
  }

  hipDeviceSynchronize();
  hipProfilerStop();
  hipStreamDestroy(stream);
  hipHostFree(ret);
  hipFree(dev_mem);
  hipFree(dev_mem2);
  hipDeviceReset();
}

void cuda_brute(worker_param_t params, uint32_t threadid, uint32_t cuda_device, int rounds) {
  uint32_t *ret = NULL;
  int32_t *dev_mem = NULL;
  hipStream_t stream = NULL;

  hipError_t err = hipSetDevice(cuda_device);
  CUDA_FAST_RETURN_ON_ERROR(err, stream, ret, dev_mem);

  /* Allocate host memory. */
  err = hipHostAlloc(&ret, sizeof(int32_t) * 2000, hipHostMallocDefault);
  CUDA_FAST_RETURN_ON_ERROR(err, stream, ret, dev_mem);
  assert(ret != NULL);

  /* Allocate device memory. */
  err = hipMalloc(&dev_mem, sizeof(int32_t) * 2000);
  CUDA_FAST_RETURN_ON_ERROR(err, stream, ret, dev_mem);
  assert(dev_mem != NULL);

  err = hipStreamCreate(&stream);
  CUDA_FAST_RETURN_ON_ERROR(err, stream, ret, dev_mem);

  uint32_t k12;
  pair_t *pair;
  int32_t hi_key_bits[24];
  int32_t pt1_bits[24];
  int32_t ct1_bits[24];
  int32_t tw1_bits[64];

  while (!g_exit && get_next_678(threadid, &k12, &pair)) {
    for (int k3 = 0; !g_exit && k3 < 0x100; k3++) {
      uint64_t k123 = (k12 << 8) | k3;
      uint32_t pt1p = enc_one_round_3(pair->t1.pt, k123 ^ (pair->t1.tw >> 40));
      uint32_t ct1p = pair->t1.ct;
      if (rounds == 8 || rounds == 15) {
        int shift = rounds == 8 ? 0 : 24;
        ct1p = dec_one_round_3(pair->t1.ct, k123 ^ ((pair->t1.tw >> shift) & 0xffffff));
      }
      uint32_t ca = ct1p >> 16;
      uint32_t cb = (ct1p >> 8) & 0xff;
      uint32_t cc = ct1p & 0xff;
      uint32_t pb = g_sbox_dec[cb] ^ ca ^ cc;
      uint32_t pc = g_sbox_dec[cc];
      uint32_t pa = g_sbox_dec[ca];
      ct1p = (pa << 16) | (pb << 8) | pc;

      /* Set plaintext, tweak and high key bits. */
      for (int bit = 0; bit < 24; bit++) {
        hi_key_bits[bit] = 0 - ((k123 >> bit) & 1);
        pt1_bits[bit]    = 0 - ((pt1p >> bit) & 1);
        ct1_bits[bit]    = 0 - ((ct1p >> bit) & 1);
      }
      for (int bit = 0; bit < 64; bit++) {
        tw1_bits[bit] = 0 - ((pair->t1.tw >> bit) & 1);
      }

      err = hipMemcpyToSymbolAsync(HIP_SYMBOL(key_c), hi_key_bits, sizeof(int32_t) * 24, 0,
          hipMemcpyHostToDevice, stream);
      CUDA_FAST_RETURN_ON_ERROR(err, stream, ret, dev_mem);
      err = hipMemcpyToSymbolAsync(HIP_SYMBOL(pt1_c), pt1_bits,    sizeof(int32_t) * 24, 0,
          hipMemcpyHostToDevice, stream);
      CUDA_FAST_RETURN_ON_ERROR(err, stream, ret, dev_mem);
      err = hipMemcpyToSymbolAsync(HIP_SYMBOL(ct1_c), ct1_bits,    sizeof(int32_t) * 24, 0,
          hipMemcpyHostToDevice, stream);
      CUDA_FAST_RETURN_ON_ERROR(err, stream, ret, dev_mem);
      err = hipMemcpyToSymbolAsync(HIP_SYMBOL(tw1_c), tw1_bits,    sizeof(int32_t) * 64, 0,
          hipMemcpyHostToDevice, stream);
      CUDA_FAST_RETURN_ON_ERROR(err, stream, ret, dev_mem);

      err = hipMemsetAsync(dev_mem, 0, sizeof(int32_t) * 2000, stream);
      CUDA_FAST_RETURN_ON_ERROR(err, stream, ret, dev_mem);

      switch (rounds) {
        case 6:
          brute_force<6><<<0x20000, 1024, 0, stream>>>(dev_mem, 0);
          break;
        case 7:
          brute_force<7><<<0x20000, 1024, 0, stream>>>(dev_mem, 0);
          break;
        case 8:
          brute_force<7><<<0x20000, 1024, 0, stream>>>(dev_mem, 0);
          break;
        case 9:
          brute_force<9><<<0x20000, 1024, 0, stream>>>(dev_mem, 0);
          break;
        case 10:
          brute_force<10><<<0x20000, 1024, 0, stream>>>(dev_mem, 0);
          break;
        case 11:
          brute_force<11><<<0x20000, 1024, 0, stream>>>(dev_mem, 0);
          break;
        case 12:
          brute_force<12><<<0x20000, 1024, 0, stream>>>(dev_mem, 0);
          break;
        case 13:
          brute_force<13><<<0x20000, 1024, 0, stream>>>(dev_mem, 0);
          break;
        case 14:
          brute_force<14><<<0x20000, 1024, 0, stream>>>(dev_mem, 0);
          break;
        case 15:
          brute_force<14><<<0x20000, 1024, 0, stream>>>(dev_mem, 0);
          break;
        case 16:
          brute_force<16><<<0x20000, 1024, 0, stream>>>(dev_mem, 0);
          break;
      }

      err = hipMemcpyAsync(ret, dev_mem, sizeof(int32_t) * 2000, hipMemcpyDefault, stream);
      CUDA_FAST_RETURN_ON_ERROR(err, stream, ret, dev_mem);

      err = hipStreamSynchronize(stream);
      CUDA_FAST_RETURN_ON_ERROR(err, stream, ret, dev_mem);

      uint64_t bkey = k123 << 32;
      for (int i = 0; i < ret[0]; i++) {
        uint64_t key = bkey | (ret[i * 2 + 1] << 5);
        uint32_t xx = ~ret[i * 2 + 2];
        while (xx != 0) {
          uint64_t lkey = (__builtin_ffs(xx) - 1) | key;
          if (test_key(params.nrounds, lkey, params.tuples, params.num_tuples)) {
            found_key(lkey);
          }
          xx ^= 1 << (lkey & 0x1f);
        }
      }
    }
  }

  hipDeviceSynchronize();
  hipProfilerStop();
  hipStreamDestroy(stream);
  hipHostFree(ret);
  hipFree(dev_mem);
  hipDeviceReset();
}
